/**
 * @file bin_pipeline_proto.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-10-10
 *
 * (C) 2022 by Indiana University, Argonne National Laboratory
 *
 */

#include <cstdio>
#include <iostream>
#include <numeric>
#include <string>

using std::cout;
using std::endl;

#include "cusz.h"
#include "hf/hf.hh"
#include "hf/hf_bookg.hh"
#include "hf/hf_codecg.hh"
#include "hf/hf_struct.h"
#include "kernel/l23.hh"
#include "kernel/lproto.hh"
#include "kernel/spv_gpu.hh"
#include "mem/layout_cxx.hh"
#include "stat/stat.hh"
#include "utils/cuda_err.cuh"
#include "utils/io.hh"
#include "utils/print_gpu.hh"
#include "utils/viewer.hh"

namespace alpha {

struct {
  uint32_t x, y, z;
} len3;

typedef struct config {
  double eb;
  int radius;
} config;

typedef struct header_superset {
  cusz_header header;
  int nnz;
  size_t total_nbit;
  size_t total_ncell;
  int hf_pardeg;
  int hf_sublen;
} header_superset;

struct hf_set {
  hf_book* book_desc;
  // hf_chunk*     chunk_desc_d;
  // hf_chunk*     chunk_desc_h;
  hf_bitstream* bitstream_desc;

  uint8_t* revbook;
  size_t revbook_nbyte;
  uint8_t* out;
  size_t outlen;

  uint8_t* hl_comp;  // high level encoder API
  size_t hl_complen;
};

// decpreated, use rt_config.h func instead
template <typename H>
int reversebook_nbyte(int booklen)
{
  return sizeof(H) * (2 * sizeof(H) * 8) + sizeof(H) * booklen;
}

}  // namespace alpha

template <
    typename T, typename E = uint32_t, typename FP = T, typename H = uint32_t,
    typename M = uint32_t>
cusz_error_status allocate_data(
    dim3 len3, alpha::hf_set* hf, cusz::HuffmanCodec<E, H, M>* codec,
    alpha::config* config)
{
  auto x = len3.x, y = len3.y, z = len3.z;
  size_t len = x * y * z;

  // coarse-grained
  auto sublen = 768;
  auto pardeg = (len + sublen - 1) / sublen;
  cout << "pardeg\t" << pardeg << endl;
  hf->book_desc->booklen = config->radius * 2;
  hf->bitstream_desc->sublen = sublen;
  hf->bitstream_desc->pardeg = pardeg;
  hf->revbook_nbyte = alpha::reversebook_nbyte<H>(hf->book_desc->booklen);

  codec->init(len, hf->book_desc->booklen, pardeg);

  CHECK_CUDA(hipMalloc(
      &hf->book_desc->freq, sizeof(uint32_t) * hf->book_desc->booklen));
  CHECK_CUDA(hipMalloc(&hf->hl_comp, 2 * len));

  // hf->d_metadata)
  return cusz_error_status::CUSZ_SUCCESS;
}

template <
    typename T, typename E = uint32_t, typename FP = T, typename H = uint32_t,
    typename M = uint32_t>
cusz_error_status deallocate_data(
    pszmempool_cxx<T, E, H>* mem, alpha::hf_set* hf,
    // cusz::HuffmanCodec<E, H, M>* codec,
    alpha::config* config)
{
  delete mem->od, delete mem->xd, delete mem;

  CHECK_CUDA(hipFree(hf->book_desc->freq));
  CHECK_CUDA(hipFree(hf->hl_comp));

  return cusz_error_status::CUSZ_SUCCESS;
}

template <
    typename T, typename E, typename FP, typename H = uint32_t,
    typename M = uint32_t>
cusz_error_status compressor(
    pszmempool_cxx<T, E, H>* mem, alpha::hf_set* hf,
    cusz::HuffmanCodec<E, H, M>* codec, alpha::config* config,
    alpha::header_superset* header_st, bool use_proto, hipStream_t stream)
{
  float time_pq = 0, time_hist = 0, time_spv = 0;
  // , _time_book = 0, time_encoding = 0;

  if (not use_proto) {
    cout << "using optimized comp. kernel\n";
    psz_comp_l23<T, E, FP>(
        mem->od->dptr(), mem->od->template len3<dim3>(), config->eb,
        config->radius, mem->ectrl_lrz(), mem->outlier_space(), &time_pq,
        stream);
  }
  else {
    cout << "using prototype comp. kernel\n";
    // psz_comp_lproto<T, E>(                                              //
    //     data->oridata->dptr(), data->len3, config->eb, config->radius,  //
    //     data->errctrl->dptr(), data->outlier->dptr(), &time, stream);
    throw runtime_error("prototype is disabled");
  }

  cout << "time-eq\t" << time_pq << endl;

  // TODO better namesapce to specify this is a firewall
  psz::spv_gather<T, M>(
      mem->outlier_space(), mem->len, mem->outlier_val(), mem->outlier_idx(),
      &header_st->nnz, &time_spv, stream);

  cout << "time-spv\t" << time_spv << endl;
  cout << "nnz\t" << header_st->nnz << endl;

  psz::stat::histogram<psz_policy::CUDA, E>(
      mem->ectrl_lrz(), mem->len, hf->book_desc->freq, hf->book_desc->booklen,
      &time_hist, stream);

  cout << "time-hist\t" << time_hist << endl;

  codec->build_codebook(hf->book_desc->freq, hf->book_desc->booklen, stream);
  codec->encode(
      mem->ectrl_lrz(), mem->len, &hf->hl_comp, &hf->hl_complen, stream);

  return cusz_error_status::CUSZ_SUCCESS;
}

template <
    typename T, typename E, typename FP, typename H = uint32_t,
    typename M = uint32_t>
cusz_error_status decompressor(
    pszmempool_cxx<T, E, H>* mem, alpha::hf_set* hf,
    cusz::HuffmanCodec<E, H, M>* codec, alpha::header_superset* header_st,
    bool use_proto, hipStream_t stream)
{
  float time_scatter = 0,
        // time_hf = 0,
      time_d_pq = 0;

  psz::spv_scatter<T, uint32_t>(
      mem->outlier_val(), mem->outlier_idx(), header_st->nnz, mem->xd->dptr(),
      &time_scatter, stream);
  cout << "decomp-time-spv\t" << time_scatter << endl;

  codec->decode(hf->hl_comp, mem->ectrl_lrz());

  if (not use_proto) {
    cout << "using optimized comp. kernel\n";
    psz_decomp_l23<T, E, FP>(
        mem->ectrl_lrz(), mem->od->template len3<dim3>(), mem->xd->dptr(),
        header_st->header.eb, header_st->header.radius, mem->xd->dptr(),
        &time_d_pq, stream);
  }
  else {
    cout << "using prototype comp. kernel\n";
    // psz_decomp_lproto<T, E, FP>(                      //
    //     data->errq, data->len3, data->outlier_, data->outlier_idx, 0,  //
    //     input header_st->header.eb, header_st->header.radius, // input
    //     (config) data->xdata, // output &time_d_pq, stream);
    throw runtime_error("prototype is disabled to later fix");
  }

  cout << "decomp-time-pq\t" << time_d_pq << endl;

  return cusz_error_status::CUSZ_SUCCESS;
}

template <typename T, typename E>
void f(
    std::string& fname, dim3 len3, alpha::hf_set* hf,
    alpha::header_superset* header_st, alpha::config* config, bool use_proto)
{
  using FP = T;
  using M = uint32_t;
  using H = uint32_t;

  hipStream_t stream;
  hipStreamCreate(&stream);

  auto mem =
      new pszmempool_cxx<T, E, H>(len3.x, config->radius, len3.y, len3.z);

  cusz::HuffmanCodec<E, uint32_t, uint32_t> codec;

  allocate_data<T, E, FP, H, M>(len3, hf, &codec, config);
  mem->od->control({Malloc, MallocHost})
      ->file(fname.c_str(), FromFile)
      ->control({H2D});
  mem->xd->control({Malloc, MallocHost});

  compressor<T, E, FP, H, M>(
      mem, hf, &codec, config, header_st, use_proto, stream);

  decompressor<T, E, FP, H, M>(mem, hf, &codec, header_st, use_proto, stream);

  /* view quality */ psz::eval_dataquality_gpu(
      mem->xd->dptr(), mem->od->dptr(), mem->len);

  deallocate_data<T, E, FP, H, M>(mem, hf, config);

  hipStreamDestroy(stream);
}

int main(int argc, char** argv)
{
  if (argc < 6) {
    printf("0    1             2     3 4 5 6  [7]     [8:128]  [9:yes]\n");
    printf(
        "PROG /path/to/file DType X Y Z EB [EType] [Radius] [Use "
        "Prototype]\n");
    printf(" 2  DType: \"F\" for `float`, \"D\" for `double`\n");
    printf(
        "[7] EType: \"ui{8,16,32}\" for `uint{8,16,32}_t` as quant-code "
        "type\n");
    exit(0);
  }

  auto fname = std::string(argv[1]);
  auto dtype = std::string(argv[2]);
  auto x = atoi(argv[3]);
  auto y = atoi(argv[4]);
  auto z = atoi(argv[5]);
  auto eb = atof(argv[6]);

  std::string etype;
  if (argc > 7)
    etype = std::string(argv[7]);
  else
    etype = "ui16";

  int radius;
  if (argc > 8)
    radius = atoi(argv[8]);
  else
    radius = 128;

  bool use_proto;
  if (argc > 9)
    use_proto = std::string(argv[9]) == "yes";
  else
    use_proto = false;

  using T = float;
  using E = uint16_t;

  auto len3 = dim3(x, y, z);

  auto header_st = new alpha::header_superset;
  header_st->header.eb = eb;
  header_st->header.radius = radius;

  auto config = new alpha::config{eb, radius};

  auto hf = new alpha::hf_set;
  hf->book_desc = new hf_book;
  hf->bitstream_desc = new hf_bitstream;
  //// dispatch

  auto radius_legal = [&](int const sizeof_T) {
    size_t upper_bound = 1lu << (sizeof_T * 8);
    cout << upper_bound << endl;
    cout << radius * 2 << endl;
    if ((radius * 2) > upper_bound)
      throw std::runtime_error("Radius overflows error-quantization type.");
  };

  // 23-06-04 restricted to u4 for quantization code

  if (dtype == "F") {
    // if (etype == "ui8") {
    //     radius_legal(1);
    //     f<float, uint8_t>(fname, len3, hf, header_st, config, use_proto);
    // }
    // else if (etype == "ui16") {
    //     radius_legal(2);
    //     f<float, uint16_t>(fname, len3, hf, header_st, config, use_proto);
    // }
    // else if (etype == "ui32") {
    // }
    radius_legal(4);
    f<float, uint32_t>(fname, len3, hf, header_st, config, use_proto);
  }
  else if (dtype == "D") {
    // if (etype == "ui8") {
    //     radius_legal(1);
    //     f<double, uint8_t>(fname, len3, hf, header_st, config, use_proto);
    // }
    // else if (etype == "ui16") {
    //     radius_legal(2);
    //     f<double, uint16_t>(fname, len3, hf, header_st, config, use_proto);
    // }
    // else if (etype == "ui32") {
    // }
    radius_legal(4);
    f<double, uint32_t>(fname, len3, hf, header_st, config, use_proto);
  }
  else
    throw std::runtime_error("not a valid dtype.");

  return 0;
}
