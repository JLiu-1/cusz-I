/**
 * @file cusz-cli.cu
 * @author Jiannan Tian
 * @brief Driver program of cuSZ.
 * @version 0.1
 * @date 2020-09-20
 * (created) 2019-12-30 (rev) 2022-02-20
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include "app.cuh"

/*
namespace {

template <typename T>
void check_shell_calls(string cmd_string)
{
    char* cmd = new char[cmd_string.length() + 1];
    strcpy(cmd, cmd_string.c_str());
    int status = system(cmd);
    delete[] cmd;
    cmd = nullptr;
    if (status < 0) { LOGGING(LOG_ERR, "Shell command call failed, exit code: ", errno, "->", strerror(errno)); }
}

}  // namespace

template <typename T, int DownscaleFactor, int tBLK>
T* pre_binning(T* d, size_t* dim_array)
{
    throw std::runtime_error("[pre_binning] disabled temporarily, will be part of preprocessing.");
    return nullptr;
}

void sparsitypath_spline3(cuszCTX* ctx)
{
    // TODO
}
*/

int main(int argc, char** argv)
{
    auto ctx = new cuszCTX(argc, argv);

    if (ctx->verbose) {
        GetMachineProperties();
        GetDeviceProperty();
    }

    cusz::app<float> cusz_cli;
    cusz_cli.cusz_dispatch(ctx);

    // if (ctx->str_predictor == "lorenzo") defaultpath(ctx);
    // if (ctx->str_predictor == "spline3") sparsitypath_spline3(ctx);
}
