#include "hip/hip_runtime.h"
/**
 * @file cusz.cu
 * @author Jiannan Tian
 * @brief Driver program of cuSZ.
 * @version 0.1
 * @date 2020-09-20
 * Created on 2019-12-30
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <math.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <cmath>
#include <cstddef>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <string>
#include <type_traits>
#include <unordered_map>
#include <vector>

using std::string;

#include "analysis/analyzer.hh"
#include "argparse.hh"
#include "capsule.hh"
#include "header.hh"
#include "kernel/preprocess.cuh"
#include "metadata.hh"
#include "nvgpusz.cuh"
#include "query.hh"
#include "type_aliasing.hh"
#include "types.hh"
#include "utils.hh"

// double expectedErr;
// double actualAbsErr;
// double actualRelErr;
// string z_mode;

namespace {

template <typename Data>
void check_shell_calls(string cmd_string)
{
    char* cmd = new char[cmd_string.length() + 1];
    strcpy(cmd, cmd_string.c_str());
    int status = system(cmd);
    delete[] cmd;
    cmd = nullptr;
    if (status < 0) { logging(log_err, "Shell command call failed, exit code: ", errno, "->", strerror(errno)); }
}

}  // namespace

/* gtest disabled in favor of code refactoring */
// TEST(cuSZTest, TestMaxError)
// {
//     double actualErr = (z_mode == "r2r") ? actualRelErr : actualAbsErr;
//     ASSERT_LE(actualErr, expectedErr);
// }

template <typename Data, int DownscaleFactor, int tBLK>
Data* pre_binning(Data* d, size_t* dim_array)
{
    throw std::runtime_error("[pre_binning] disabled temporarily, will be part of preprocessing.");
    return nullptr;
}

#define NONPTR_TYPE(VAR) std::remove_pointer<decltype(VAR)>::type

int main(int argc, char** argv)
{
    auto ctx = new ArgPack();
    ctx->parse_args(argc, argv);

    if (ctx->verbose) {
        GetMachineProperties();
        GetDeviceProperty();
    }

    // TODO remove hardcode for float for now
    using Data = float;

    auto len = ctx->data_len;
    auto m   = static_cast<size_t>(ceil(sqrt(len)));
    auto mxm = m * m;

    Capsule<Data> in_data(mxm);

    if (ctx->task_is.construct or ctx->task_is.dryrun) {
        // logging(log_dbg, "add padding:", m, "units");

        hipMalloc(&in_data.dptr, in_data.nbyte());
        hipHostMalloc(&in_data.hptr, in_data.nbyte());

        {
            auto a = hires::now();
            io::read_binary_to_array<Data>(ctx->fnames.path2file, in_data.hptr, len);
            auto z = hires::now();

            if (ctx->verbose) logging(log_dbg, "time loading datum:", static_cast<duration_t>(z - a).count(), "sec");

            logging(log_info, "load", ctx->fnames.path2file, len * sizeof(Data), "bytes");
        }

        in_data.h2d();

        if (ctx->mode == "r2r") {
            Analyzer analyzer;
            auto     result = analyzer.GetMaxMinRng                                     //
                          <Data, ExecutionPolicy::cuda_device, AnalyzerMethod::thrust>  //
                          (in_data.dptr, len);
            if (ctx->verbose) logging(log_dbg, "time scanning:", result.seconds, "sec");
            ctx->eb *= result.rng;
        }

        if (ctx->verbose)
            logging(
                log_dbg, std::to_string(ctx->quant_nbyte) + "-byte quant type,",
                std::to_string(ctx->huff_nbyte) + "-byte internal Huff type");
    }

    if (ctx->task_is.pre_binning) {
        cerr << log_err
             << "Binning is not working temporarily; we are improving end-to-end throughput by NOT touching "
                "filesystem. (ver. 0.1.4)"
             << endl;
        exit(1);
    }

    if (ctx->task_is.construct or ctx->task_is.dryrun) {  // fp32 only for now

        if (ctx->quant_nbyte == 1) {
            throw runtime_error("Quant=1-byte temporarily disabled.");
            if (ctx->huff_nbyte == 4) {
                // cusz_compress<true, 4, 1, 4>(ctx, &in_data);
            }
            else {
                // cusz_compress<true, 4, 1, 8>(ctx, &in_data);
            }
        }
        else if (ctx->quant_nbyte == 2) {
            if (ctx->huff_nbyte == 4) {  //
                cusz_compress<true, 4, 2, 4>(ctx, &in_data);
            }
            else {
                cusz_compress<true, 4, 2, 8>(ctx, &in_data);
            }
        }

        // release memory
        hipFree(in_data.dptr), hipHostFree(in_data.hptr);
    }

    if (in_data.dptr) {
        hipHostFree(in_data.dptr);  // TODO messy
    }

    if (ctx->task_is.reconstruct) {  // fp32 only for now

        // TODO data ready outside Decompressor?

        if (ctx->quant_nbyte == 1) {
            throw runtime_error("Quant=1-byte temporarily disabled.");
            if (ctx->huff_nbyte == 4) {
                // cusz_decompress<true, 4, 1, 4>(ctx);
            }
            else if (ctx->huff_nbyte == 8) {
                // cusz_decompress<true, 4, 1, 8>(ctx);
            }
        }
        else if (ctx->quant_nbyte == 2) {
            if (ctx->huff_nbyte == 4)
                cusz_decompress<true, 4, 2, 4>(ctx);
            else if (ctx->huff_nbyte == 8)
                cusz_decompress<true, 4, 2, 8>(ctx);
        }
    }
}
