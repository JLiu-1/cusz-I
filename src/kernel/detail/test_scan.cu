/**
 * @file test_scan.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.4
 * @date 2022-12-23
 *
 * (C) 2022 by Indiana University, Argonne National Laboratory
 *
 */

#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <string>

using std::cout;
using std::endl;
using std::string;

#include "../../../test/src/rand.hh"
#include "lorenzo.inl"
#include "lorenzo23.inl"

template <int BLOCK = 256, int SEQ = 8>
void test_inclusive_scan()
{
    using T  = float;
    using EQ = uint16_t;
    using FP = T;

    constexpr auto NTHREAD = BLOCK / SEQ;

    auto len  = BLOCK;
    auto ebx2 = 1;

    T*  data{nullptr};
    EQ* eq{nullptr};

    hipMallocManaged(&data, sizeof(T) * len);
    hipMallocManaged(&eq, sizeof(EQ) * len);
    hipMemset(eq, 0x0, sizeof(EQ) * len);

    {
        cout << "original" << endl;
        for (auto i = 0; i < BLOCK; i++) data[i] = 1;

        cusz::x_lorenzo_1d1l<T, EQ, FP, BLOCK, SEQ>
            <<<1, NTHREAD>>>(data, eq, data, dim3(len, 1, 1), dim3(0, 0, 0), 0, ebx2);
        hipDeviceSynchronize();

        for (auto i = 0; i < BLOCK; i++) cout << data[i] << " ";
        cout << "\n" << endl;
    }

    {
        cout << "refactored v0 (wave32)" << endl;
        for (auto i = 0; i < BLOCK; i++) data[i] = 1;

        parsz::cuda::__kernel::v0::x_lorenzo_1d1l<T, EQ, FP, BLOCK, SEQ>
            <<<1, NTHREAD>>>(eq, data, dim3(len, 1, 1), dim3(0, 0, 0), 0, ebx2, data);
        hipDeviceSynchronize();

        for (auto i = 0; i < BLOCK; i++) cout << data[i] << " ";
        cout << "\n" << endl;
    }

    hipFree(data);
    hipFree(eq);
}

int main()
{
    test_inclusive_scan<256, 4>();
    test_inclusive_scan<256, 8>();
    test_inclusive_scan<512, 4>();
    test_inclusive_scan<512, 8>();
    test_inclusive_scan<1024, 4>();
    test_inclusive_scan<1024, 8>();

    return 0;
}