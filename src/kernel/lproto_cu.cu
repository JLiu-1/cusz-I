/**
 * @file lproto.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-09-22
 *
 * (C) 2022 by Indiana University, Argonne National Laboratory
 *
 */

#include "cusz/type.h"
#include "detail/lproto.inl"
#include "kernel/lproto.hh"
#include "mem/compact.hh"
#include "utils/err.hh"
#include "utils/timer.hh"

template <typename T, typename Eq>
pszerror psz_comp_lproto(
    T* const data, dim3 const len3, double const eb, int const radius,
    Eq* const eq, void* _outlier, float* time_elapsed, hipStream_t stream)
{
  auto divide3 = [](dim3 len, dim3 sublen) {
    return dim3(
        (len.x - 1) / sublen.x + 1, (len.y - 1) / sublen.y + 1,
        (len.z - 1) / sublen.z + 1);
  };

  auto ndim = [&]() {
    if (len3.z == 1 and len3.y == 1)
      return 1;
    else if (len3.z == 1 and len3.y != 1)
      return 2;
    else
      return 3;
  };

  auto outlier = (CompactGpuDram<T>*)_outlier;

  constexpr auto Tile1D = 256;
  constexpr auto Block1D = dim3(256, 1, 1);
  auto Grid1D = divide3(len3, Tile1D);

  constexpr auto Tile2D = dim3(16, 16, 1);
  constexpr auto Block2D = dim3(16, 16, 1);
  auto Grid2D = divide3(len3, Tile2D);

  constexpr auto Tile3D = dim3(8, 8, 8);
  constexpr auto Block3D = dim3(8, 8, 8);
  auto Grid3D = divide3(len3, Tile3D);

  // error bound
  auto ebx2 = eb * 2;
  auto ebx2_r = 1 / ebx2;
  auto leap3 = dim3(1, len3.x, len3.x * len3.y);

  CREATE_GPUEVENT_PAIR;
  START_GPUEVENT_RECORDING(stream);

  using namespace psz::cuda_hip::__kernel::proto;

  if (ndim() == 1) {
    c_lorenzo_1d1l<T, Eq><<<Grid1D, Block1D, 0, stream>>>(
        data, len3, leap3, radius, ebx2_r, eq, *outlier);
  }
  else if (ndim() == 2) {
    c_lorenzo_2d1l<T, Eq><<<Grid2D, Block2D, 0, stream>>>(
        data, len3, leap3, radius, ebx2_r, eq, *outlier);
  }
  else if (ndim() == 3) {
    c_lorenzo_3d1l<T, Eq><<<Grid3D, Block3D, 0, stream>>>(
        data, len3, leap3, radius, ebx2_r, eq, *outlier);
  }
  else {
    throw std::runtime_error("Lorenzo only works for 123-D.");
  }

  STOP_GPUEVENT_RECORDING(stream);
  CHECK_GPU(hipStreamSynchronize(stream));

  TIME_ELAPSED_GPUEVENT(time_elapsed);
  DESTROY_GPUEVENT_PAIR;

  return CUSZ_SUCCESS;
}

template <typename T, typename Eq>
pszerror psz_decomp_lproto(
    Eq* eq, dim3 const len3, T* scattered_outlier, double const eb,
    int const radius, T* xdata, float* time_elapsed, hipStream_t stream)
{
  auto divide3 = [](dim3 len, dim3 sublen) {
    return dim3(
        (len.x - 1) / sublen.x + 1, (len.y - 1) / sublen.y + 1,
        (len.z - 1) / sublen.z + 1);
  };

  auto ndim = [&]() {
    if (len3.z == 1 and len3.y == 1)
      return 1;
    else if (len3.z == 1 and len3.y != 1)
      return 2;
    else
      return 3;
  };

  constexpr auto Tile1D = 256;
  constexpr auto Block1D = dim3(256, 1, 1);
  auto Grid1D = divide3(len3, Tile1D);

  constexpr auto Tile2D = dim3(16, 16, 1);
  constexpr auto Block2D = dim3(16, 16, 1);
  auto Grid2D = divide3(len3, Tile2D);

  constexpr auto Tile3D = dim3(8, 8, 8);
  constexpr auto Block3D = dim3(8, 8, 8);
  auto Grid3D = divide3(len3, Tile3D);

  // error bound
  auto ebx2 = eb * 2;
  auto ebx2_r = 1 / ebx2;
  auto leap3 = dim3(1, len3.x, len3.x * len3.y);

  CREATE_GPUEVENT_PAIR;
  START_GPUEVENT_RECORDING(stream);

  using namespace psz::cuda_hip::__kernel::proto;

  if (ndim() == 1) {
    x_lorenzo_1d1l<T, Eq><<<Grid1D, Block1D, 0, stream>>>(
        eq, scattered_outlier, len3, leap3, radius, ebx2, xdata);
  }
  else if (ndim() == 2) {
    x_lorenzo_2d1l<T, Eq><<<Grid2D, Block2D, 0, stream>>>(
        eq, scattered_outlier, len3, leap3, radius, ebx2, xdata);
  }
  else if (ndim() == 3) {
    x_lorenzo_3d1l<T, Eq><<<Grid3D, Block3D, 0, stream>>>(
        eq, scattered_outlier, len3, leap3, radius, ebx2, xdata);
  }

  STOP_GPUEVENT_RECORDING(stream);
  CHECK_GPU(hipStreamSynchronize(stream));

  TIME_ELAPSED_GPUEVENT(time_elapsed);
  DESTROY_GPUEVENT_PAIR;

  return CUSZ_SUCCESS;
}

#define CPP_INS(T, Eq)                                                 \
  template pszerror psz_comp_lproto<T, Eq>(                   \
      T* const, dim3 const, double const, int const, Eq* const, void*, \
      float*, hipStream_t);                                           \
                                                                       \
  template pszerror psz_decomp_lproto<T, Eq>(                 \
      Eq*, dim3 const, T*, double const, int const, T*, float*,        \
      hipStream_t);

// TODO decrease the number of instantiated types
CPP_INS(float, uint8_t);
CPP_INS(float, uint16_t);
CPP_INS(float, uint32_t);
// CPP_INS(float, float);
// CPP_INS(float, int32_t);

CPP_INS(double, uint8_t);
CPP_INS(double, uint16_t);
CPP_INS(double, uint32_t);
// CPP_INS(double, float);
// CPP_INS(double, int32_t);

#undef CPP_INS
