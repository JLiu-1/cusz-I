/**
 * @file timer_gpu.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-10-31
 *
 * (C) 2022 by Indiana University, Argonne National Laboratory
 *
 */

#include <hip/hip_runtime.h>
#include "utils/timer.h"

// cuda timer specific

asz_timer* asz_cudatimer_create()
{
    auto t = new asz_timer{
        .policy = asz_policy::CUDA,  //
        .start  = new hipEvent_t,
        .end    = new hipEvent_t};

    hipEventCreate(static_cast<hipEvent_t*>(t->start));
    hipEventCreate(static_cast<hipEvent_t*>(t->end));

    return t;
}

void asz_cudatimer_destroy(asz_timer* t)
{
    delete static_cast<hipEvent_t*>(t->start);
    delete static_cast<hipEvent_t*>(t->end);
    delete t;
}

void asz_cudatimer_start(asz_timer* t)
{  //
    hipEventRecord(*static_cast<hipEvent_t*>(t->start));
}

void asz_cudatimer_end(asz_timer* t)
{  //
    hipEventRecord(*static_cast<hipEvent_t*>(t->end));
    hipEventSynchronize(*static_cast<hipEvent_t*>(t->end));
}

double asz_cudatime_elapsed(asz_timer* t)
{
    float second;
    hipEventElapsedTime(&second, *static_cast<hipEvent_t*>(t->start), *static_cast<hipEvent_t*>(t->end));
    return second / 1000;
}

// cuda streamtimer specific

asz_timer* asz_cudastreamtimer_create(void* stream)
{
    auto t = new asz_timer{
        .policy = asz_policy::CUDA,  //
        .start  = new hipEvent_t,
        .end    = new hipEvent_t,
        .stream = stream};

    hipEventCreate(static_cast<hipEvent_t*>(t->start));
    hipEventCreate(static_cast<hipEvent_t*>(t->end));

    return t;
}

void asz_cudastreamtimer_destroy(asz_timer* t)
{
    delete static_cast<hipEvent_t*>(t->start);
    delete static_cast<hipEvent_t*>(t->end);
    delete t;
}

void asz_cudastreamtimer_start(asz_timer* t)
{  //
    hipEventRecord(*static_cast<hipEvent_t*>(t->start), static_cast<hipStream_t>(t->stream));
}

void asz_cudastreamtimer_end(asz_timer* t)
{  //
    hipEventRecord(*static_cast<hipEvent_t*>(t->end), static_cast<hipStream_t>(t->stream));
    hipEventSynchronize(*static_cast<hipEvent_t*>(t->end));
}

double asz_cudastreamtime_elapsed(asz_timer* t)
{
    float second;
    hipEventElapsedTime(&second, *static_cast<hipEvent_t*>(t->start), *static_cast<hipEvent_t*>(t->end));
    return second / 1000;
}
