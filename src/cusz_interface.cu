/**
 * @file cusz_workflow.cu
 * @author Jiannan Tian
 * @brief Workflow of cuSZ.
 * @version 0.3
 * @date 2021-07-12
 * (create) 2020-02-12; (release) 2020-09-20; (rev.1) 2021-01-16; (rev.2) 2021-07-12
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cxxabi.h>
#include <bitset>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <type_traits>
#include <typeinfo>

#include "analysis/analyzer.hh"
#include "argparse.hh"
#include "cusz_interface.h"
#include "kernel/dryrun.h"
#include "kernel/lorenzo.h"
#include "metadata.hh"
#include "type_trait.hh"
#include "utils.hh"
#include "wrapper/extrap_lorenzo.h"
#include "wrapper/handle_sparsity.h"
#include "wrapper/huffman_enc_dec.cuh"
#include "wrapper/huffman_parbook.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;

#define DATATYPE struct PartialData<typename DataTrait<If_FP, DataByte>::Data>

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz_compress(argpack* ap, DATATYPE* in_data, dim3 xyz, metadata_pack* mp, unsigned int optional_w)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    Compressor<Data, Quant, Huff, float> cuszc(ap, ap->len, ap->eb);

    cuszc.lorenzo_dryrun(in_data);  // subject to change

    struct PartialData<Quant> quant(ap->len + HuffConfig::Db_encode);
    hipMalloc(&quant.dptr, quant.nbyte());

    struct PartialData<unsigned int> freq(ap->dict_size);
    hipMalloc(&freq.dptr, freq.nbyte());

    struct PartialData<Huff> book(ap->dict_size);
    hipMalloc(&book.dptr, book.nbyte()), book.memset(0xff);

    struct PartialData<uint8_t> revbook(cuszc.get_revbook_nbyte());
    hipMalloc(&revbook.dptr, revbook.nbyte());
    hipHostMalloc(&revbook.hptr, revbook.nbyte());  // to write to disk later

    cuszc  //
        .predict_quantize(in_data, xyz, &quant)
        .gather_outlier(in_data)
        .try_skip_huffman(&quant);

    // release in_data; subject to change
    hipFree(in_data->dptr);

    cuszc.get_freq_and_codebook(&quant, &freq, &book, &revbook)
        .analyze_compressibility(&freq, &book)
        .internal_eval_try_export_book(&book)
        .internal_eval_try_export_quant(&quant)
        .export_revbook(&revbook)
        .huffman_encode(&quant, &book)
        .try_report_time()
        .pack_metadata(mp);

    hipFree(quant.dptr), hipFree(freq.dptr), hipFree(book.dptr), hipFree(revbook.dptr);
}

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz_decompress(argpack* ap, metadata_pack* mp)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    Decompressor<Data, Quant, Huff, float> cuszd(mp, ap);

    auto xyz = dim3(ap->dim4._0, ap->dim4._1, ap->dim4._2);

    struct PartialData<Quant> quant(cuszd.length.quant);
    hipMalloc(&quant.dptr, quant.nbyte());
    hipHostMalloc(&quant.hptr, quant.nbyte());

    struct PartialData<Data> _data(cuszd.mxm + MetadataTrait<1>::Block);  // TODO ad hoc size
    hipMalloc(&_data.dptr, _data.nbyte());
    hipHostMalloc(&_data.hptr, _data.nbyte());
    auto xdata = _data.dptr, outlier = _data.dptr;

    cuszd.huffman_decode(&quant)
        .scatter_outlier(outlier)
        .reversed_predict_quantize(xdata, quant.dptr, xyz)
        .try_report_time();

    // copy decompressed data to host
    _data.d2h();

    cuszd
        .calculate_archive_nbyte()  //
        .try_compare(_data.hptr)
        .try_write2disk(_data.hptr);
}

#define CUSZ_COMPRESS(DBYTE, QBYTE, HBYTE)                  \
    template void cusz_compress<true, DBYTE, QBYTE, HBYTE>( \
        argpack*, struct PartialData<float>*, dim3, metadata_pack*, unsigned int);

CUSZ_COMPRESS(4, 1, 4)
CUSZ_COMPRESS(4, 1, 8)
CUSZ_COMPRESS(4, 2, 4)
CUSZ_COMPRESS(4, 2, 8)

#define CUSZ_DECOMPRESS(DBYTE, QBYTE, HBYTE) \
    template void cusz_decompress<true, DBYTE, QBYTE, HBYTE>(argpack*, metadata_pack*);

CUSZ_DECOMPRESS(4, 1, 4)
CUSZ_DECOMPRESS(4, 1, 8)
CUSZ_DECOMPRESS(4, 2, 4)
CUSZ_DECOMPRESS(4, 2, 8)
