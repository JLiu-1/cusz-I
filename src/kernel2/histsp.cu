/**
 * @file hist_sp.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.4
 * @date 2023-05-18
 *
 * (C) 2023 by Indiana University, Argonne National Laboratory
 *
 */

#include <cstdint>

#include "detail2/histsp.inl"
#include "kernel2/histsp.hh"

namespace psz {
namespace detail {

template <typename T, typename FQ>
int histsp_cuda(
    T* in, uint32_t inlen, FQ* out_hist, uint32_t outlen, hipStream_t stream)
{
  constexpr auto CHUNK = 32768;
  constexpr auto NWARP = 8;
  constexpr auto NTREAD = 32 * NWARP;

  histsp_multiwarp<T, NWARP, CHUNK, FQ>
      <<<(inlen - 1) / CHUNK + 1, NTREAD, sizeof(FQ) * outlen, stream>>>(
          in, inlen, out_hist, outlen, outlen / 2);

  return 0;
}

}  // namespace detail
}  // namespace psz

// template int histsp_cuda<uint32_t>(
//     uint32_t*, uint32_t, uint32_t*, uint32_t, hipStream_t);

template <>
int histsp<psz_policy::CUDA, uint32_t, uint32_t>(
    uint32_t* in, uint32_t inlen, uint32_t* out_hist, uint32_t outlen,
    void* stream)
{
  return psz::detail::histsp_cuda<uint32_t, uint32_t>(
      in, inlen, out_hist, outlen, (hipStream_t)stream);
}
