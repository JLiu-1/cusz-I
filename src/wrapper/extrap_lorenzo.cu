/**
 * @file extrap_lorenzo.cu
 * @author Jiannan Tian
 * @brief A high-level LorenzoND wrapper. Allocations are explicitly out of called functions.
 * @version 0.3
 * @date 2021-06-16
 * (rev.1) 2021-09-18
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include <iostream>
#include <limits>
#include <numeric>
#include <stdexcept>

#include "../common.hh"
#include "../utils.hh"

#include "extrap_lorenzo.cuh"

#ifdef DPCPP_SHOWCASE
#include "../kernel/lorenzo_prototype.cuh"

using cusz::prototype::c_lorenzo_1d1l;
using cusz::prototype::c_lorenzo_2d1l;
using cusz::prototype::c_lorenzo_3d1l;
using cusz::prototype::x_lorenzo_1d1l;
using cusz::prototype::x_lorenzo_2d1l;
using cusz::prototype::x_lorenzo_3d1l;

#else
#include "../kernel/lorenzo.cuh"
#endif

#if __cplusplus >= 201703L
#define CONSTEXPR constexpr
#else
#define CONSTEXPR
#endif

namespace {

#ifndef __HIPCC__
struct __dim3_compat {
    unsigned int x, y, z;
    __dim3_compat(unsigned int _x, unsigned int _y, unsigned int _z){};
};

using dim3 = __dim3_compat;
#endif

}  // namespace

template <typename T, typename E, typename FP>
cusz::PredictorLorenzo<T, E, FP>::PredictorLorenzo(dim3 xyz, bool _delay_postquant)
{
    // size
    size      = xyz;
    leap      = dim3(1, size.x, size.x * size.y);
    len_data  = size.x * size.y * size.z;
    len_quant = len_data;

    len_outlier = len_data;

    ndim = 3;
    if (size.z == 1) ndim = 2;
    if (size.z == 1 and size.y == 1) ndim = 1;

    // on off
    delay_postquant = _delay_postquant;
}

template <typename T, typename E, typename FP>
template <bool DELAY_POSTQUANT>
void cusz::PredictorLorenzo<T, E, FP>::construct_proxy(
    T* const in_data,
    T* const out_anchor,
    E* const out_errctrl,
    T* const __restrict__ __out_outlier,
    double const       eb,
    int const          radius,
    hipStream_t const stream)
{
    // error bound
    auto ebx2   = eb * 2;
    auto ebx2_r = 1 / ebx2;

    // decide if destructive for the input (data)
    auto out_outlier = __out_outlier == nullptr ? in_data : __out_outlier;

    // TODO put into conditional compile
    cuda_timer_t timer;
    timer.timer_start(stream);

    if (ndim == 1) {
        constexpr auto SEQ          = 4;
        constexpr auto DATA_SUBSIZE = 256;
        auto           dim_block    = DATA_SUBSIZE / SEQ;
        auto           dim_grid     = ConfigHelper::get_npart(size.x, DATA_SUBSIZE);
        cusz::c_lorenzo_1d1l<T, E, FP, DATA_SUBSIZE, SEQ, DELAY_POSTQUANT>  //
            <<<dim_grid, dim_block, 0, stream>>>                            //
            (in_data, out_errctrl, out_outlier, size.x, radius, ebx2_r);
    }
    else if (ndim == 2) {  // y-sequentiality == 8
        auto dim_block = dim3(16, 2);
        auto dim_grid  = dim3(ConfigHelper::get_npart(size.x, 16), ConfigHelper::get_npart(size.y, 16));
        cusz::c_lorenzo_2d1l_16x16data_mapto16x2<T, E, FP>  //
            <<<dim_grid, dim_block, 0, stream>>>            //
            (in_data, out_errctrl, out_outlier, size.x, size.y, leap.y, radius, ebx2_r);
    }
    else if (ndim == 3) {  // y-sequentiality == 8
        auto dim_block = dim3(32, 1, 8);
        auto dim_grid  = dim3(
             ConfigHelper::get_npart(size.x, 32), ConfigHelper::get_npart(size.y, 8),
             ConfigHelper::get_npart(size.z, 8));
        cusz::c_lorenzo_3d1l_32x8x8data_mapto32x1x8<T, E, FP>  //
            <<<dim_grid, dim_block, 0, stream>>>               //
            (in_data, out_errctrl, out_outlier, size.x, size.y, size.z, leap.y, leap.z, radius, ebx2_r);
    }
    else {
        throw std::runtime_error("Lorenzo only works for 123-D.");
    }

    timer.timer_end(stream);
    if (stream)
        CHECK_CUDA(hipStreamSynchronize(stream));
    else
        CHECK_CUDA(hipDeviceSynchronize());

    time_elapsed = timer.get_time_elapsed();
}

template <typename T, typename E, typename FP>
template <bool DELAY_POSTQUANT>
void cusz::PredictorLorenzo<T, E, FP>::reconstruct_proxy(
    T* const __restrict__ __in_outlier,
    T* const           in_anchor,
    E* const           in_errctrl,
    T* const           out_xdata,
    double const       eb,
    int const          radius,
    hipStream_t const stream)
{
    // error bound
    auto ebx2   = eb * 2;
    auto ebx2_r = 1 / ebx2;

    // decide if destructive for the input (outlier)
    auto in_outlier = __in_outlier == nullptr ? out_xdata : __in_outlier;

    cuda_timer_t timer;
    timer.timer_start(stream);

    if (ndim == 1) {  // y-sequentiality == 8
        constexpr auto SEQ          = 8;
        constexpr auto DATA_SUBSIZE = 256;
        auto           dim_block    = DATA_SUBSIZE / SEQ;
        auto           dim_grid     = ConfigHelper::get_npart(size.x, DATA_SUBSIZE);
        cusz::x_lorenzo_1d1l<T, E, FP, DATA_SUBSIZE, SEQ, DELAY_POSTQUANT>  //
            <<<dim_grid, dim_block, 0, stream>>>                            //
            (in_outlier, in_errctrl, out_xdata, size.x, radius, ebx2);
    }
    else if (ndim == 2) {  // y-sequentiality == 8
        auto dim_block = dim3(16, 2);
        auto dim_grid  = dim3(ConfigHelper::get_npart(size.x, 16), ConfigHelper::get_npart(size.y, 16));

        cusz::x_lorenzo_2d1l_16x16data_mapto16x2<T, E, FP, DELAY_POSTQUANT>  //
            <<<dim_grid, dim_block, 0, stream>>>                             //
            (in_outlier, in_errctrl, out_xdata, size.x, size.y, leap.y, radius, ebx2);
    }
    else if (ndim == 3) {  // y-sequentiality == 8
        auto dim_block = dim3(32, 1, 8);
        auto dim_grid  = dim3(
             ConfigHelper::get_npart(size.x, 32), ConfigHelper::get_npart(size.y, 8),
             ConfigHelper::get_npart(size.z, 8));

        cusz::x_lorenzo_3d1l_32x8x8data_mapto32x1x8<T, E, FP, DELAY_POSTQUANT>  //
            <<<dim_grid, dim_block, 0, stream>>>                                //
            (in_outlier, in_errctrl, out_xdata, size.x, size.y, size.z, leap.y, leap.z, radius, ebx2);
    }

    timer.timer_end(stream);
    if (stream)
        CHECK_CUDA(hipStreamSynchronize(stream));
    else
        CHECK_CUDA(hipDeviceSynchronize());

    time_elapsed = timer.get_time_elapsed();
}

template <typename T, typename E, typename FP>
void cusz::PredictorLorenzo<T, E, FP>::construct(
    T* const           in_data,
    T* const           out_anchor,
    E* const           out_errctrl,
    double const       eb,
    int const          radius,
    hipStream_t const stream,
    T* __restrict__ non_overlap_out_outlier)
{
    if (not delay_postquant)
        construct_proxy<false>(in_data, out_anchor, out_errctrl, non_overlap_out_outlier, eb, radius, stream);
    else
        throw std::runtime_error("construct_proxy<delay_postquant==true> not implemented.");
}

template <typename T, typename E, typename FP>
void cusz::PredictorLorenzo<T, E, FP>::reconstruct(
    T* const           in_anchor,
    E* const           in_errctrl,
    T* const           out_xdata,
    double const       eb,
    int const          radius,
    hipStream_t const stream,
    T* __restrict__ non_overlap_in_outlier)
{
    if (not delay_postquant)
        reconstruct_proxy<false>(non_overlap_in_outlier, in_anchor, in_errctrl, out_xdata, eb, radius, stream);
    else
        throw std::runtime_error("construct_proxy<delay_postquant==true> not implemented.");
}

template class cusz::PredictorLorenzo<float, uint16_t, float>;
template class cusz::PredictorLorenzo<float, float, float>;