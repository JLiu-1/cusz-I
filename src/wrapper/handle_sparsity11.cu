/**
 * @file handle_sparsity11.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2021-09-28
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <cstddef>
#include <iostream>

#include "../common.hh"
#include "../utils.hh"

#include "handle_sparsity11.cuh"

using handle_t = hipsparseHandle_t;
using stream_t = hipStream_t;
// using descr_t  = hipsparseMatDescr_t;

/********************************************************************************
 * compression use
 ********************************************************************************/

namespace cusz {

template <typename T>
OutlierHandler11<T>::OutlierHandler11(unsigned int _len, unsigned int* init_workspace_nbyte)
{
    if (init_workspace_nbyte == nullptr)
        throw std::runtime_error("[OutlierHandler11::constructor] init_workspace_nbyte must not be null.");

    m = Reinterpret1DTo2D::get_square_size(_len);

    // TODO merge to configure?
    auto initial_nnz = _len / SparseMethodSetup::factor;
    // set up pool
    offset.rowptr = 0;
    offset.colidx = sizeof(int) * (m + 1);
    offset.values = sizeof(int) * (m + 1) + sizeof(int) * initial_nnz;

    *init_workspace_nbyte = sizeof(int) * (m + 1) +      // rowptr
                            sizeof(int) * initial_nnz +  // colidx
                            sizeof(T) * initial_nnz;     // values
}

template <typename T>
void OutlierHandler11<T>::configure_workspace(uint8_t* _pool)
{
    if (not _pool) throw std::runtime_error("Memory is no allocated.");
    pool_ptr     = _pool;
    entry.rowptr = reinterpret_cast<int*>(pool_ptr + offset.rowptr);
    entry.colidx = reinterpret_cast<int*>(pool_ptr + offset.colidx);
    entry.values = reinterpret_cast<T*>(pool_ptr + offset.values);
}

template <typename T>
void OutlierHandler11<T>::reconfigure_with_precise_nnz(int nnz)
{
    this->nnz    = nnz;
    nbyte.rowptr = sizeof(int) * (m + 1);
    nbyte.colidx = sizeof(int) * nnz;
    nbyte.values = sizeof(T) * nnz;
    nbyte.total  = nbyte.rowptr + nbyte.colidx + nbyte.values;
}

template <typename T>
void OutlierHandler11<T>::gather_CUDA11(T* in_data, unsigned int& _dump_poolsize)
{
    hipsparseHandle_t     handle = nullptr;
    hipsparseSpMatDescr_t matB;  // sparse
    hipsparseDnMatDescr_t matA;  // dense
    void*                dBuffer    = nullptr;
    size_t               bufferSize = 0;

    auto d_dense = in_data;

    CHECK_CUSPARSE(hipsparseCreate(&handle));

    auto num_rows = m;
    auto num_cols = m;
    auto ld       = m;

    // Create dense matrix A
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&matA, num_rows, num_cols, ld, d_dense, cuszCUSPARSE<T>::type, HIPSPARSE_ORDER_ROW));

    // Create sparse matrix B in CSR format
    auto d_csr_offsets = entry.rowptr;
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matB, num_rows, num_cols, 0, d_csr_offsets, nullptr, nullptr, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, cuszCUSPARSE<T>::type));

    // allocate an external buffer if needed
    {
        auto t = new cuda_timer_t;
        t->timer_start();

        CHECK_CUSPARSE(
            hipsparseDenseToSparse_bufferSize(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;

        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    }

    // execute Sparse to Dense conversion
    {
        auto t = new cuda_timer_t;
        t->timer_start();

        CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;
    }

    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, __nnz;
    /**  this is all HOST, skip timing **/
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp, &__nnz));

    auto d_csr_columns = entry.colidx;
    auto d_csr_values  = entry.values;

    // allocate CSR column indices and values (skipped in customiztion)

    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE(hipsparseCsrSetPointers(matB, d_csr_offsets, d_csr_columns, d_csr_values));

    // execute Sparse to Dense conversion
    {
        auto t = new cuda_timer_t;
        t->timer_start();

        CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;
    }

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    /********************************************************************************/
    reconfigure_with_precise_nnz(__nnz);
    dump_nbyte     = query_csr_bytelen();
    _dump_poolsize = dump_nbyte;
}

template <typename T>
void OutlierHandler11<T>::archive(uint8_t* dst, int& export_nnz, hipMemcpyKind direction)
{
    export_nnz = this->nnz;

    // clang-format off
    hipMemcpy(dst + 0,                           entry.rowptr, nbyte.rowptr, direction);
    hipMemcpy(dst + nbyte.rowptr,                entry.colidx, nbyte.colidx, direction);
    hipMemcpy(dst + nbyte.rowptr + nbyte.colidx, entry.values, nbyte.values, direction);
    // clang-format on
}

/********************************************************************************
 * decompression use
 ********************************************************************************/

template <typename T>
OutlierHandler11<T>::OutlierHandler11(unsigned int _len, unsigned int _nnz)
{  //
    this->m   = Reinterpret1DTo2D::get_square_size(_len);
    this->nnz = _nnz;

    nbyte.rowptr = sizeof(int) * (this->m + 1);
    nbyte.colidx = sizeof(int) * this->nnz;
    nbyte.values = sizeof(T) * this->nnz;
    nbyte.total  = nbyte.rowptr + nbyte.colidx + nbyte.values;
}

template <typename T>
void OutlierHandler11<T>::extract(uint8_t* _pool)
{
    offset.rowptr = 0;
    offset.colidx = nbyte.rowptr;
    offset.values = nbyte.rowptr + nbyte.colidx;

    pool_ptr     = _pool;
    entry.rowptr = reinterpret_cast<int*>(pool_ptr + offset.rowptr);
    entry.colidx = reinterpret_cast<int*>(pool_ptr + offset.colidx);
    entry.values = reinterpret_cast<T*>(pool_ptr + offset.values);
};

template <typename T>
void OutlierHandler11<T>::scatter_CUDA11(T* out_dn)
{
    auto d_csr_offsets = entry.rowptr;
    auto d_csr_columns = entry.colidx;
    auto d_csr_values  = entry.values;

    /********************************************************************************/

    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;

    auto num_rows = m;
    auto num_cols = m;
    auto ld       = m;

    auto d_dense = out_dn;

    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, num_rows, num_cols, nnz, d_csr_offsets, d_csr_columns, d_csr_values, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, cuszCUSPARSE<T>::type));
    // Create dense matrix B
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&matB, num_rows, num_cols, ld, d_dense, cuszCUSPARSE<T>::type, HIPSPARSE_ORDER_ROW));

    {
        auto t = new cuda_timer_t;
        t->timer_start();

        // allocate an external buffer if needed
        CHECK_CUSPARSE(
            hipsparseSparseToDense_bufferSize(handle, matA, matB, HIPSPARSE_SPARSETODENSE_ALG_DEFAULT, &bufferSize));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;
    }
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // execute Sparse to Dense conversion
    {
        auto t = new cuda_timer_t;
        t->timer_start();

        CHECK_CUSPARSE(hipsparseSparseToDense(handle, matA, matB, HIPSPARSE_SPARSETODENSE_ALG_DEFAULT, dBuffer));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;
    }

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
}

//
}  // namespace cusz

template class cusz::OutlierHandler11<float>;
