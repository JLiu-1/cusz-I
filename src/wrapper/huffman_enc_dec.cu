#include "hip/hip_runtime.h"
/**
 * @file huffman_enc_dec.cu
 * @author Jiannan Tian, Cody Rivera (cjrivera1@crimson.ua.edu)
 * @brief Workflow of Huffman coding.
 * @version 0.1
 * @date 2020-10-24
 * (created) 2020-04-24 (rev) 2021-09-0
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>

#include <sys/stat.h>
#include <unistd.h>
#include <algorithm>
#include <bitset>
#include <cassert>
#include <cmath>
#include <functional>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <string>
#include <tuple>
#include <type_traits>
#include <vector>

#include "../common.hh"
#include "../kernel/codec_huffman.cuh"
#include "../kernel/hist.cuh"
#include "../utils.hh"
#include "huffman_enc_dec.cuh"

#ifdef MODULAR_ELSEWHERE
#include "cascaded.hpp"
#include "nvcomp.hpp"
#endif

#if __cplusplus >= 201703L
#define CONSTEXPR constexpr
#else
#define CONSTEXPR
#endif

#define nworker blockDim.x

template <typename Huff>
__global__ void cusz::huffman_enc_concatenate(
    Huff*   in_enc_space,
    Huff*   out_bitstream,
    size_t* sp_entries,
    size_t* sp_uints,
    size_t  chunk_size)
{
    auto len      = sp_uints[blockIdx.x];
    auto sp_entry = sp_entries[blockIdx.x];
    auto dn_entry = chunk_size * blockIdx.x;

    for (auto i = 0; i < (len + nworker - 1) / nworker; i++) {
        auto _tid = threadIdx.x + i * nworker;
        if (_tid < len) *(out_bitstream + sp_entry + _tid) = *(in_enc_space + dn_entry + _tid);
        __syncthreads();
    }
}

template <typename Huff>
void cusz::huffman_process_metadata(
    size_t* _counts,
    size_t* dev_bits,
    size_t  nchunk,
    size_t& num_bits,
    size_t& num_uints)
{
    constexpr auto TYPE_BITCOUNT = sizeof(Huff) * 8;

    auto sp_uints = _counts, sp_bits = _counts + nchunk, sp_entries = _counts + nchunk * 2;

    hipMemcpy(sp_bits, dev_bits, nchunk * sizeof(size_t), hipMemcpyDeviceToHost);
    memcpy(sp_uints, sp_bits, nchunk * sizeof(size_t));
    for_each(sp_uints, sp_uints + nchunk, [&](size_t& i) { i = (i + TYPE_BITCOUNT - 1) / TYPE_BITCOUNT; });
    memcpy(sp_entries + 1, sp_uints, (nchunk - 1) * sizeof(size_t));
    for (auto i = 1; i < nchunk; i++) sp_entries[i] += sp_entries[i - 1];  // inclusive scan

    num_bits  = std::accumulate(sp_bits, sp_bits + nchunk, (size_t)0);
    num_uints = std::accumulate(sp_uints, sp_uints + nchunk, (size_t)0);
}

/*
template <typename T>
void draft::UseNvcompZip(T* space, size_t& len)
{
    int*         uncompressed_data;
    const size_t in_bytes = len * sizeof(T);

    hipMalloc(&uncompressed_data, in_bytes);
    hipMemcpy(uncompressed_data, space, in_bytes, hipMemcpyHostToDevice);
    hipStream_t stream;
    hipStreamCreate(&stream);
    // 2 layers RLE, 1 Delta encoding, bitpacking enabled
    nvcomp::CascadedCompressor<int> compressor(uncompressed_data, in_bytes / sizeof(int), 2, 1, true);
    const size_t                    temp_size = compressor.get_temp_size();
    void*                           temp_space;
    hipMalloc(&temp_space, temp_size);
    size_t output_size = compressor.get_max_output_size(temp_space, temp_size);
    void*  output_space;
    hipMalloc(&output_space, output_size);
    compressor.compress_async(temp_space, temp_size, output_space, &output_size, stream);
    hipStreamSynchronize(stream);
    // TODO ad hoc; should use original GPU space
    memset(space, 0x0, len * sizeof(T));
    len = output_size / sizeof(T);
    hipMemcpy(space, output_space, output_size, hipMemcpyDeviceToHost);

    hipFree(uncompressed_data);
    hipFree(temp_space);
    hipFree(output_space);
    hipStreamDestroy(stream);
}

template <typename T>
void draft::UseNvcompUnzip(T** d_space, size_t& len)
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    nvcomp::Decompressor<int> decompressor(*d_space, len * sizeof(T), stream);
    const size_t              temp_size = decompressor.get_temp_size();
    void*                     temp_space;
    hipMalloc(&temp_space, temp_size);

    const size_t output_count = decompressor.get_num_elements();
    int*         output_space;
    hipMalloc((void**)&output_space, output_count * sizeof(int));

    decompressor.decompress_async(temp_space, temp_size, output_space, output_count, stream);

    hipStreamSynchronize(stream);
    hipFree(*d_space);

    *d_space = mem::create_CUDA_space<T>((unsigned long)(output_count * sizeof(int)));
    hipMemcpy(*d_space, output_space, output_count * sizeof(int), hipMemcpyDeviceToDevice);
    len = output_count * sizeof(int) / sizeof(T);

    hipFree(output_space);

    hipStreamDestroy(stream);
    hipFree(temp_space);
}

*/

template <typename Quant, typename Huff, bool UINTS_KNOWN>
void lossless::HuffmanEncode(
    Huff*   dev_enc_space,
    size_t* dev_bits,
    size_t* dev_uints,
    size_t* dev_entries,
    size_t* host_counts,
    //
    Huff* dev_out_bitstream,
    //
    Quant*  dev_input,
    Huff*   dev_book,
    size_t  len,
    int     chunk_size,
    int     dict_size,
    size_t* ptr_num_bits,
    size_t* ptr_num_uints,
    float&  milliseconds)
{
    auto nchunk = ConfigHelper::get_npart(len, chunk_size);

    if CONSTEXPR (UINTS_KNOWN == false) {
        {
            auto block_dim = HuffmanHelper::BLOCK_DIM_ENCODE;
            auto grid_dim  = ConfigHelper::get_npart(len, block_dim);

            int numSMs;
            hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

            LOGGING(LOG_WARN, "#SM: ", numSMs);
            LOGGING(LOG_WARN, "len: ", len);

            auto t = new cuda_timer_t;
            t->timer_start();
            // cusz::encode_fixedlen_space_cub<Quant, Huff, HuffmanHelper::ENC_SEQUENTIALITY>
            //     <<<grid_dim, block_dim / HuffmanHelper::ENC_SEQUENTIALITY>>>(dev_input, dev_enc_space, len,
            //     dev_book);

            cusz::encode_fixedlen_gridstride        //
                <Quant, Huff><<<8 * numSMs, 256>>>  //
                (dev_input, dev_enc_space, len, dev_book, dict_size);
            milliseconds += t->timer_end_get_elapsed_time();
            CHECK_CUDA(hipDeviceSynchronize());
            delete t;
        }

        {
            auto block_dim = HuffmanHelper::BLOCK_DIM_DEFLATE;
            auto grid_dim  = ConfigHelper::get_npart(nchunk, block_dim);
            auto t         = new cuda_timer_t;
            t->timer_start();
            cusz::encode_deflate<Huff><<<grid_dim, block_dim>>>(dev_enc_space, len, dev_bits, chunk_size);
            milliseconds += t->timer_end_get_elapsed_time();
            hipDeviceSynchronize();
            delete t;
        }

        cusz::huffman_process_metadata<Huff>(host_counts, dev_bits, nchunk, *ptr_num_bits, *ptr_num_uints);
        hipMemcpy(dev_uints, host_counts, nchunk * sizeof(size_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_entries, (host_counts + nchunk * 2), nchunk * sizeof(size_t), hipMemcpyHostToDevice);
    }
    else {
        auto t = new cuda_timer_t;
        t->timer_start();
        cusz::huffman_enc_concatenate<<<nchunk, 128>>>(
            dev_enc_space, dev_out_bitstream, dev_entries, dev_uints, chunk_size);
        milliseconds += t->timer_end_get_elapsed_time();
        hipDeviceSynchronize();
    }
}

// TODO mark types using Q/H-byte binding; internally resolve UI8-UI8_2 issue

#define HUFFMAN_ENCODE(Q, H, BOOL)                     \
    template void lossless::HuffmanEncode<Q, H, BOOL>( \
        H*, size_t*, size_t*, size_t*, size_t*, H*, Q*, H*, size_t, int, int, size_t*, size_t*, float&);

HUFFMAN_ENCODE(ErrCtrlTrait<2>::type, HuffTrait<4>::type, false)
HUFFMAN_ENCODE(ErrCtrlTrait<2>::type, HuffTrait<8>::type, false)
HUFFMAN_ENCODE(ErrCtrlTrait<4>::type, HuffTrait<4>::type, false)
HUFFMAN_ENCODE(ErrCtrlTrait<4>::type, HuffTrait<8>::type, false)

HUFFMAN_ENCODE(ErrCtrlTrait<2>::type, HuffTrait<4>::type, true)
HUFFMAN_ENCODE(ErrCtrlTrait<2>::type, HuffTrait<8>::type, true)
HUFFMAN_ENCODE(ErrCtrlTrait<4>::type, HuffTrait<4>::type, true)
HUFFMAN_ENCODE(ErrCtrlTrait<4>::type, HuffTrait<8>::type, true)
