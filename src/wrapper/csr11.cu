/**
 * @file csr11.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2021-09-28
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <cstddef>
#include <iostream>

#include "../common.hh"
#include "../utils.hh"

#include "csr11.cuh"

using handle_t = hipsparseHandle_t;
using stream_t = hipStream_t;
// using descr_t  = hipsparseMatDescr_t;

/********************************************************************************
 * compression use
 ********************************************************************************/

namespace cusz {

template <typename T>
void CSR11<T>::reconfigure_with_precise_nnz(int nnz)
{
    this->nnz    = nnz;
    nbyte.rowptr = sizeof(int) * (m + 1);
    nbyte.colidx = sizeof(int) * nnz;
    nbyte.values = sizeof(T) * nnz;
    nbyte.total  = nbyte.rowptr + nbyte.colidx + nbyte.values;
}

#if CUDART_VERSION >= 11000

template <typename T>
void CSR11<T>::gather_CUDA11(T* in_data, unsigned int& _dump_poolsize)
{
    hipsparseHandle_t     handle = nullptr;
    hipsparseSpMatDescr_t matB;  // sparse
    hipsparseDnMatDescr_t matA;  // dense
    void*                dBuffer    = nullptr;
    size_t               bufferSize = 0;

    auto d_dense = in_data;

    CHECK_CUSPARSE(hipsparseCreate(&handle));

    auto num_rows = m;
    auto num_cols = m;
    auto ld       = m;

    // Create dense matrix A
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&matA, num_rows, num_cols, ld, d_dense, cuszCUSPARSE<T>::type, HIPSPARSE_ORDER_ROW));

    // Create sparse matrix B in CSR format
    auto d_csr_offsets = rowptr.template get<DEFAULT_LOC>();
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matB, num_rows, num_cols, 0, d_csr_offsets, nullptr, nullptr, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, cuszCUSPARSE<T>::type));

    // allocate an external buffer if needed
    {
        auto t = new cuda_timer_t;
        t->timer_start();

        CHECK_CUSPARSE(
            hipsparseDenseToSparse_bufferSize(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;

        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    }

    // execute Sparse to Dense conversion
    {
        auto t = new cuda_timer_t;
        t->timer_start();

        CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;
    }

    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, __nnz;
    /**  this is all HOST, skip timing **/
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp, &__nnz));

    auto d_csr_columns = colidx.template get<DEFAULT_LOC>();
    auto d_csr_values  = values.template get<DEFAULT_LOC>();

    // allocate CSR column indices and values (skipped in customiztion)

    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE(hipsparseCsrSetPointers(matB, d_csr_offsets, d_csr_columns, d_csr_values));

    // execute Sparse to Dense conversion
    {
        auto t = new cuda_timer_t;
        t->timer_start();

        CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;
    }

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    /********************************************************************************/
    reconfigure_with_precise_nnz(__nnz);
    dump_nbyte     = query_csr_bytelen();
    _dump_poolsize = dump_nbyte;
}

#elif CUDART_VERSION >= 10000

template <typename T>
void CSR11<T>::gather_CUDA10(T* in_outlier, unsigned int& _dump_poolsize)
{
    hipsparseHandle_t   handle       = nullptr;
    hipStream_t       stream       = nullptr;
    hipsparseMatDescr_t mat_desc     = nullptr;
    size_t             lworkInBytes = 0;
    char*              d_work       = nullptr;
    float              threshold    = 0;
    auto               n            = m;
    auto               lda          = m;

    // clang-format off
    CHECK_CUDA(hipStreamCreateWithFlags   ( &stream,    hipStreamNonBlocking        )); // 1. create stream
    CHECK_CUSPARSE(hipsparseCreate          ( &handle                                  )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream       (  handle,    stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &mat_desc                                )); // 4. create mat_desc
    CHECK_CUSPARSE(hipsparseSetMatIndexBase (  mat_desc,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType      (  mat_desc,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type
    // clang-format on

    {
        auto timer_step3 = new cuda_timer_t;
        timer_step3->timer_start();

        CHECK_CUSPARSE(hipsparseSpruneDense2csr_bufferSizeExt(  //
            handle, m, n, in_outlier, lda, &threshold, mat_desc, values.template get<DEFAULT_LOC>(),
            rowptr.template get<DEFAULT_LOC>(), colidx.template get<DEFAULT_LOC>(), &lworkInBytes));

        milliseconds += timer_step3->timer_end_get_elapsed_time();
        delete timer_step3;
    }

    if (nullptr != d_work) hipFree(d_work);
    CHECK_CUDA(hipMalloc((void**)&d_work, lworkInBytes));  // TODO where to release d_work?

    auto nnz = 0;

    /* step 4: compute rowptr and nnz */
    {
        auto timer_step4 = new cuda_timer_t;
        timer_step4->timer_start();

        CHECK_CUSPARSE(hipsparseSpruneDense2csrNnz(  //
            handle, m, n, in_outlier, lda, &threshold, mat_desc, rowptr.template get<DEFAULT_LOC>(), &nnz, d_work));

        milliseconds += timer_step4->timer_end_get_elapsed_time();
        CHECK_CUDA(hipDeviceSynchronize());
        delete timer_step4;
    }

    reconfigure_with_precise_nnz(nnz);

    if (nnz == 0) {
        std::cout << "nnz == 0, exiting gather.\n";
        // return *this;
        return;
    }

    /* step 5: compute col_ind and values */
    {
        auto timer_step5 = new cuda_timer_t;
        timer_step5->timer_start();

        CHECK_CUSPARSE(hipsparseSpruneDense2csr(  //
            handle, m, n, in_outlier, lda, &threshold, mat_desc, values.template get<DEFAULT_LOC>(),
            rowptr.template get<DEFAULT_LOC>(), colidx.template get<DEFAULT_LOC>(), d_work));

        milliseconds += timer_step5->timer_end_get_elapsed_time();
        CHECK_CUDA(hipDeviceSynchronize());
        delete timer_step5;
    }

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (mat_desc) hipsparseDestroyMatDescr(mat_desc);

    /********************************************************************************/
    dump_nbyte     = query_csr_bytelen();
    _dump_poolsize = dump_nbyte;
    /********************************************************************************/
}

#else
#error CUDART_VERSION must be no less than 10.0!
#endif

template <typename T>
template <cusz::LOC FROM, cusz::LOC TO>
CSR11<T>& CSR11<T>::consolidate(uint8_t* dst)
{
    constexpr auto direction = CopyDirection<FROM, TO>::direction;
    // clang-format off
    hipMemcpy(dst + 0,                           rowptr.template get<DEFAULT_LOC>(), nbyte.rowptr, direction);
    hipMemcpy(dst + nbyte.rowptr,                colidx.template get<DEFAULT_LOC>(), nbyte.colidx, direction);
    hipMemcpy(dst + nbyte.rowptr + nbyte.colidx, values.template get<DEFAULT_LOC>(), nbyte.values, direction);
    // clang-format on
    return *this;
}

template <typename T>
CSR11<T>& CSR11<T>::decompress_set_nnz(unsigned int _nnz)
{  //
    this->nnz = _nnz;

    nbyte.rowptr = sizeof(int) * (this->m + 1);
    nbyte.colidx = sizeof(int) * this->nnz;
    nbyte.values = sizeof(T) * this->nnz;
    nbyte.total  = nbyte.rowptr + nbyte.colidx + nbyte.values;

    return *this;
}

template <typename T>
void CSR11<T>::extract(uint8_t* _pool)
{
    offset.rowptr = 0;
    offset.colidx = nbyte.rowptr;
    offset.values = nbyte.rowptr + nbyte.colidx;

    pool_ptr                           = _pool;
    rowptr.template get<DEFAULT_LOC>() = reinterpret_cast<int*>(pool_ptr + offset.rowptr);
    colidx.template get<DEFAULT_LOC>() = reinterpret_cast<int*>(pool_ptr + offset.colidx);
    values.template get<DEFAULT_LOC>() = reinterpret_cast<T*>(pool_ptr + offset.values);
};

#if CUDART_VERSION >= 11000

template <typename T>
void CSR11<T>::scatter_CUDA11(T* out_dn)
{
    auto d_csr_offsets = rowptr.template get<DEFAULT_LOC>();
    auto d_csr_columns = colidx.template get<DEFAULT_LOC>();
    auto d_csr_values  = values.template get<DEFAULT_LOC>();

    /********************************************************************************/

    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;

    auto num_rows = m;
    auto num_cols = m;
    auto ld       = m;

    auto d_dense = out_dn;

    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, num_rows, num_cols, nnz, d_csr_offsets, d_csr_columns, d_csr_values, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, cuszCUSPARSE<T>::type));
    // Create dense matrix B
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&matB, num_rows, num_cols, ld, d_dense, cuszCUSPARSE<T>::type, HIPSPARSE_ORDER_ROW));

    {
        auto t = new cuda_timer_t;
        t->timer_start();

        // allocate an external buffer if needed
        CHECK_CUSPARSE(
            hipsparseSparseToDense_bufferSize(handle, matA, matB, HIPSPARSE_SPARSETODENSE_ALG_DEFAULT, &bufferSize));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;
    }
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // execute Sparse to Dense conversion
    {
        auto t = new cuda_timer_t;
        t->timer_start();

        CHECK_CUSPARSE(hipsparseSparseToDense(handle, matA, matB, HIPSPARSE_SPARSETODENSE_ALG_DEFAULT, dBuffer));

        milliseconds += t->timer_end_get_elapsed_time();
        delete t;
    }

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
}

#elif CUDART_VERSION >= 10000

template <typename T>
void CSR11<T>::scatter_CUDA10(T* out_dn)
{
    //     throw std::runtime_error("[decompress_scatter] not implemented");
    hipsparseHandle_t   handle   = nullptr;
    hipStream_t       stream   = nullptr;
    hipsparseMatDescr_t mat_desc = nullptr;
    auto               n        = m;
    auto               lda      = m;

    // clang-format off
    CHECK_CUDA(hipStreamCreateWithFlags   ( &stream,   hipStreamNonBlocking        )); // 1. create stream
    CHECK_CUSPARSE(hipsparseCreate          ( &handle                                 )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream       (  handle,   stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &mat_desc                               )); // 4. create descr
    CHECK_CUSPARSE(hipsparseSetMatIndexBase (  mat_desc, HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType      (  mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL )); // type
    // clang-format on

    {
        auto timer_scatter = new cuda_timer_t;
        timer_scatter->timer_start();

        CHECK_CUSPARSE(hipsparseScsr2dense(
            handle, m, n, mat_desc, values.template get<DEFAULT_LOC>(), rowptr.template get<DEFAULT_LOC>(),
            colidx.template get<DEFAULT_LOC>(), out_dn, lda));

        milliseconds += timer_scatter->timer_end_get_elapsed_time();
        CHECK_CUDA(hipDeviceSynchronize());
        delete timer_scatter;
    }

    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (mat_desc) hipsparseDestroyMatDescr(mat_desc);
}

#else
#error CUDART_VERSION must be no less than 10.0!
#endif

//
}  // namespace cusz

#define CSR11_TYPE cusz::CSR11<float>

template class CSR11_TYPE;

template CSR11_TYPE& CSR11_TYPE::consolidate<cusz::LOC::HOST, cusz::LOC::HOST>(uint8_t*);
template CSR11_TYPE& CSR11_TYPE::consolidate<cusz::LOC::HOST, cusz::LOC::DEVICE>(uint8_t*);
template CSR11_TYPE& CSR11_TYPE::consolidate<cusz::LOC::DEVICE, cusz::LOC::HOST>(uint8_t*);
template CSR11_TYPE& CSR11_TYPE::consolidate<cusz::LOC::DEVICE, cusz::LOC::DEVICE>(uint8_t*);
