/**
 * @file csr11.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2021-09-28
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <cstddef>
#include <iostream>

#include "../common.hh"
#include "../utils.hh"

#include "csr11.cuh"

/********************************************************************************
 * compression use
 ********************************************************************************/

namespace cusz {

template <typename T>
void CSR11<T>::reconfigure_with_precise_nnz(int nnz)
{
    this->nnz    = nnz;
    nbyte.rowptr = sizeof(int) * (m + 1);
    nbyte.colidx = sizeof(int) * nnz;
    nbyte.values = sizeof(T) * nnz;
    nbyte.total  = nbyte.rowptr + nbyte.colidx + nbyte.values;
}

#if CUDART_VERSION >= 11020

template <typename T>
void CSR11<T>::gather_CUDA11(T* in_data, unsigned int& _dump_poolsize, hipStream_t stream)
{
    hipsparseHandle_t     handle = nullptr;
    hipsparseSpMatDescr_t matB;  // sparse
    hipsparseDnMatDescr_t matA;  // dense
    void*                dBuffer    = nullptr;
    size_t               bufferSize = 0;

    auto d_dense = in_data;

    CHECK_CUSPARSE(hipsparseCreate(&handle));

    if (stream) CHECK_CUSPARSE(hipsparseSetStream(handle, stream));

    auto num_rows = m;
    auto num_cols = m;
    auto ld       = m;

    // Create dense matrix A
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&matA, num_rows, num_cols, ld, d_dense, cuszCUSPARSE<T>::type, HIPSPARSE_ORDER_ROW));

    // Create sparse matrix B in CSR format
    auto d_csr_offsets = rowptr.template get<DEFAULT_LOC>();
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matB, num_rows, num_cols, 0, d_csr_offsets, nullptr, nullptr, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, cuszCUSPARSE<T>::type));

    // allocate an external buffer if needed
    {
        cuda_timer_t t;
        t.timer_start(stream);

        CHECK_CUSPARSE(
            hipsparseDenseToSparse_bufferSize(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize));

        t.timer_end(stream);
        milliseconds += t.get_time_elapsed();

        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    }

    // execute Sparse to Dense conversion
    {
        cuda_timer_t t;
        t.timer_start(stream);

        CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));

        t.timer_end(stream);
        milliseconds += t.get_time_elapsed();
    }

    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, __nnz;
    /**  this is all HOST, skip timing **/
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp, &__nnz));

    auto d_csr_columns = colidx.template get<DEFAULT_LOC>();
    auto d_csr_values  = values.template get<DEFAULT_LOC>();

    // allocate CSR column indices and values (skipped in customiztion)

    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE(hipsparseCsrSetPointers(matB, d_csr_offsets, d_csr_columns, d_csr_values));

    // execute Sparse to Dense conversion
    {
        cuda_timer_t t;
        t.timer_start(stream);

        CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));

        t.timer_end(stream);
        milliseconds += t.get_time_elapsed();
    }

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    /********************************************************************************/
    reconfigure_with_precise_nnz(__nnz);
    dump_nbyte     = query_csr_bytelen();
    _dump_poolsize = dump_nbyte;
}

#elif CUDART_VERSION >= 10000

template <typename T>
void CSR11<T>::gather_CUDA10(T* in_outlier, unsigned int& _dump_poolsize, hipStream_t ext_stream)
{
    hipsparseHandle_t   handle       = nullptr;
    hipStream_t       stream       = nullptr;
    hipsparseMatDescr_t mat_desc     = nullptr;
    size_t             lworkInBytes = 0;
    char*              d_work       = nullptr;
    float              threshold    = 0;
    auto               n            = m;
    auto               lda          = m;

    auto has_ext_stream = false;

    if (ext_stream) {
        has_ext_stream = true;
        stream         = ext_stream;
    }
    else {
        CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));  // 1. create stream
    }

    // clang-format off
    CHECK_CUSPARSE(hipsparseCreate          ( &handle                                  )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream       (  handle,    stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &mat_desc                                )); // 4. create mat_desc
    CHECK_CUSPARSE(hipsparseSetMatIndexBase (  mat_desc,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType      (  mat_desc,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type
    // clang-format on

    {
        cuda_timer_t t;
        t.timer_start(stream);

        CHECK_CUSPARSE(hipsparseSpruneDense2csr_bufferSizeExt(  //
            handle, m, n, in_outlier, lda, &threshold, mat_desc, values.template get<DEFAULT_LOC>(),
            rowptr.template get<DEFAULT_LOC>(), colidx.template get<DEFAULT_LOC>(), &lworkInBytes));

        t.timer_end(stream);
        milliseconds += t.get_time_elapsed();
    }

    if (nullptr != d_work) hipFree(d_work);
    CHECK_CUDA(hipMalloc((void**)&d_work, lworkInBytes));  // TODO where to release d_work?

    auto nnz = 0;

    /* step 4: compute rowptr and nnz */
    {
        cuda_timer_t t.;
        t.timer_start(stream);

        CHECK_CUSPARSE(hipsparseSpruneDense2csrNnz(  //
            handle, m, n, in_outlier, lda, &threshold, mat_desc, rowptr.template get<DEFAULT_LOC>(), &nnz, d_work));

        t.timer_end(stream);
        milliseconds += t.get_time_elapsed();
        CHECK_CUDA(hipStreamSynchronize(stream));
    }

    reconfigure_with_precise_nnz(nnz);

    if (nnz == 0) {
        std::cout << "nnz == 0, exiting gather.\n";
        // return *this;
        return;
    }

    /* step 5: compute col_ind and values */
    {
        cuda_timer_t t;
        t.timer_start(stream);

        CHECK_CUSPARSE(hipsparseSpruneDense2csr(  //
            handle, m, n, in_outlier, lda, &threshold, mat_desc, values.template get<DEFAULT_LOC>(),
            rowptr.template get<DEFAULT_LOC>(), colidx.template get<DEFAULT_LOC>(), d_work));

        t.timer_end(stream);
        milliseconds += t.get_time_elapsed();
        CHECK_CUDA(hipStreamSynchronize(stream));
        delete timer_step5;
    }

    if (handle) hipsparseDestroy(handle);
    if (mat_desc) hipsparseDestroyMatDescr(mat_desc);

    if ((not has_ext_stream) and stream) hipStreamDestroy(stream);

    /********************************************************************************/
    dump_nbyte     = query_csr_bytelen();
    _dump_poolsize = dump_nbyte;
    /********************************************************************************/
}

#else
#error CUDART_VERSION must be no less than 10.0!
#endif

template <typename T>
template <cusz::LOC FROM, cusz::LOC TO>
CSR11<T>& CSR11<T>::consolidate(uint8_t* dst)
{
    constexpr auto direction = CopyDirection<FROM, TO>::direction;
    // clang-format off
    hipMemcpy(dst + 0,                           rowptr.template get<DEFAULT_LOC>(), nbyte.rowptr, direction);
    hipMemcpy(dst + nbyte.rowptr,                colidx.template get<DEFAULT_LOC>(), nbyte.colidx, direction);
    hipMemcpy(dst + nbyte.rowptr + nbyte.colidx, values.template get<DEFAULT_LOC>(), nbyte.values, direction);
    // clang-format on
    return *this;
}

template <typename T>
CSR11<T>& CSR11<T>::decompress_set_nnz(unsigned int _nnz)
{  //
    this->nnz = _nnz;

    nbyte.rowptr = sizeof(int) * (this->m + 1);
    nbyte.colidx = sizeof(int) * this->nnz;
    nbyte.values = sizeof(T) * this->nnz;
    nbyte.total  = nbyte.rowptr + nbyte.colidx + nbyte.values;

    return *this;
}

template <typename T>
void CSR11<T>::extract(uint8_t* _pool)
{
    offset.rowptr = 0;
    offset.colidx = nbyte.rowptr;
    offset.values = nbyte.rowptr + nbyte.colidx;

    pool_ptr                           = _pool;
    rowptr.template get<DEFAULT_LOC>() = reinterpret_cast<int*>(pool_ptr + offset.rowptr);
    colidx.template get<DEFAULT_LOC>() = reinterpret_cast<int*>(pool_ptr + offset.colidx);
    values.template get<DEFAULT_LOC>() = reinterpret_cast<T*>(pool_ptr + offset.values);
};

#if CUDART_VERSION >= 11020

template <typename T>
void CSR11<T>::scatter_CUDA11(T* out_dn, hipStream_t stream)
{
    auto d_csr_offsets = rowptr.template get<DEFAULT_LOC>();
    auto d_csr_columns = colidx.template get<DEFAULT_LOC>();
    auto d_csr_values  = values.template get<DEFAULT_LOC>();

    /********************************************************************************/

    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;

    auto num_rows = m;
    auto num_cols = m;
    auto ld       = m;

    auto d_dense = out_dn;

    CHECK_CUSPARSE(hipsparseCreate(&handle));

    if (stream) CHECK_CUSPARSE(hipsparseSetStream(handle, stream));

    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, num_rows, num_cols, nnz, d_csr_offsets, d_csr_columns, d_csr_values, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, cuszCUSPARSE<T>::type));
    // Create dense matrix B
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&matB, num_rows, num_cols, ld, d_dense, cuszCUSPARSE<T>::type, HIPSPARSE_ORDER_ROW));

    {
        cuda_timer_t t;
        t.timer_start(stream);

        // allocate an external buffer if needed
        CHECK_CUSPARSE(
            hipsparseSparseToDense_bufferSize(handle, matA, matB, HIPSPARSE_SPARSETODENSE_ALG_DEFAULT, &bufferSize));

        t.timer_end(stream);
        milliseconds += t.get_time_elapsed();
    }
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // execute Sparse to Dense conversion
    {
        cuda_timer_t t;
        t.timer_start(stream);

        CHECK_CUSPARSE(hipsparseSparseToDense(handle, matA, matB, HIPSPARSE_SPARSETODENSE_ALG_DEFAULT, dBuffer));

        t.timer_end(stream);
        milliseconds += t.get_time_elapsed();
    }

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
}

#elif CUDART_VERSION >= 10000

template <typename T>
void CSR11<T>::scatter_CUDA10(T* out_dn, hipStream_t ext_stream)
{
    hipsparseHandle_t   handle   = nullptr;  // TODO move cusparse handle outside
    hipStream_t       stream   = nullptr;
    hipsparseMatDescr_t mat_desc = nullptr;
    auto               n        = m;
    auto               lda      = m;

    auto has_external_stream = false;

    if (ext_stream) {
        has_external_stream = true;
        stream              = ext_stream;
    }
    else {
        CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));  // 1. create stream
    }

    // clang-format off
    CHECK_CUSPARSE(hipsparseCreate          ( &handle                                 )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream       (  handle,   stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &mat_desc                               )); // 4. create descr
    CHECK_CUSPARSE(hipsparseSetMatIndexBase (  mat_desc, HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType      (  mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL )); // type
    // clang-format on

    {
        cuda_timer_t t;
        t.timer_start(stream);

        CHECK_CUSPARSE(hipsparseScsr2dense(
            handle, m, n, mat_desc, values.template get<DEFAULT_LOC>(), rowptr.template get<DEFAULT_LOC>(),
            colidx.template get<DEFAULT_LOC>(), out_dn, lda));

        t.timer_end();
        milliseconds += t.get_time_elapsed();
        CHECK_CUDA(hipStreamSynchronize(stream));
    }

    // TODO move cusparse handle outside
    if (handle) hipsparseDestroy(handle);

    if (mat_desc) hipsparseDestroyMatDescr(mat_desc);

    if ((not has_external_stream) and stream) hipStreamDestroy(stream);
}

#else
#error CUDART_VERSION must be no less than 10.0!
#endif

//
}  // namespace cusz

#define CSR11_TYPE cusz::CSR11<float>

template class CSR11_TYPE;

template CSR11_TYPE& CSR11_TYPE::consolidate<cusz::LOC::HOST, cusz::LOC::HOST>(uint8_t*);
template CSR11_TYPE& CSR11_TYPE::consolidate<cusz::LOC::HOST, cusz::LOC::DEVICE>(uint8_t*);
template CSR11_TYPE& CSR11_TYPE::consolidate<cusz::LOC::DEVICE, cusz::LOC::HOST>(uint8_t*);
template CSR11_TYPE& CSR11_TYPE::consolidate<cusz::LOC::DEVICE, cusz::LOC::DEVICE>(uint8_t*);
