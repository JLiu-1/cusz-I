#include "hip/hip_runtime.h"
/**
 * @file nvgpusz.cu
 * @author Jiannan Tian
 * @brief Workflow of cuSZ.
 * @version 0.3
 * @date 2021-07-12
 * (create) 2020-02-12; (release) 2020-09-20; (rev.1) 2021-01-16; (rev.2) 2021-07-12; (rev.3) 2021-09-06
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cxxabi.h>
#include <bitset>
#include <cstdlib>
#include <exception>
#include <fstream>
#include <iostream>
#include <type_traits>
#include <typeinfo>

#include "analysis/analyzer.hh"
#include "argparse.hh"
#include "kernel/dryrun.cuh"
#include "kernel/lorenzo.cuh"
#include "metadata.hh"
#include "nvgpusz.cuh"
#include "type_trait.hh"
#include "utils.hh"
#include "wrapper/extrap_lorenzo.cuh"
#include "wrapper/handle_sparsity.cuh"
#include "wrapper/huffman_enc_dec.cuh"
#include "wrapper/huffman_parbook.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;

////////////////////////////////////////////////////////////////////////////////

#define COMPR_TYPE template <typename Data, typename Quant, typename Huff, typename FP>
#define COMPRESSOR Compressor<Data, Quant, Huff, FP>

COMPR_TYPE
unsigned int COMPRESSOR::tune_deflate_chunksize(size_t len)
{
    int current_dev = 0;
    hipSetDevice(current_dev);
    hipDeviceProp_t dev_prop{};
    hipGetDeviceProperties(&dev_prop, current_dev);

    auto nSM                = dev_prop.multiProcessorCount;
    auto allowed_block_dim  = dev_prop.maxThreadsPerBlock;
    auto deflate_nthread    = allowed_block_dim * nSM / HuffConfig::deflate_constant;
    auto optimal_chunk_size = (len + deflate_nthread - 1) / deflate_nthread;
    optimal_chunk_size      = ((optimal_chunk_size - 1) / HuffConfig::Db_deflate + 1) * HuffConfig::Db_deflate;

    return optimal_chunk_size;
}

COMPR_TYPE
void COMPRESSOR::report_compression_time(size_t len, float lossy, float outlier, float hist, float book, float lossless)
{
    auto display_throughput = [](float milliseconds, size_t nbyte) {
        auto GiB        = 1.0 * 1024 * 1024 * 1024;
        auto seconds    = milliseconds * 1e-3;
        auto throughput = nbyte / GiB / seconds;
        cout << throughput << "GiB/s\n";
    };
    //
    cout << "\nTIME in milliseconds\t================================================================\n";
    float nonbook = lossy + outlier + hist + lossless;

    printf("TIME\tconstruct:\t%f\t", lossy), display_throughput(lossy, len * sizeof(Data));
    printf("TIME\toutlier:\t%f\t", outlier), display_throughput(outlier, len * sizeof(Data));
    printf("TIME\thistogram:\t%f\t", hist), display_throughput(hist, len * sizeof(Data));
    printf("TIME\tencode:\t%f\t", lossless), display_throughput(lossless, len * sizeof(Data));

    cout << "TIME\t--------------------------------------------------------------------------------\n";
    printf("TIME\tnon-book kernels (sum):\t%f\t", nonbook), display_throughput(nonbook, len * sizeof(Data));
    cout << "TIME\t================================================================================\n";
    printf("TIME\tbuild book (not counted in prev section):\t%f\t", book), display_throughput(book, len * sizeof(Data));
    printf("TIME\t*all* kernels (sum, count book time):\t%f\t", nonbook + book),
        display_throughput(nonbook + book, len * sizeof(Data));
    cout << "TIME\t================================================================================\n\n";
}

COMPR_TYPE
COMPRESSOR::Compressor(argpack* _ctx) : ctx(_ctx)
{
    ndim = ctx->ndim;

    config.radius = ctx->radius;

    length.data      = ctx->data_len;
    length.quant     = length.data;  // TODO if lorenzo
    length.dict_size = ctx->dict_size;

    config.eb     = ctx->eb;
    config.ebx2   = ctx->eb * 2;
    config.ebx2_r = 1 / (ctx->eb * 2);
    config.eb_r   = 1 / ctx->eb;

    if (ctx->task_is.autotune_huffchunk) ctx->huffman_chunk = tune_deflate_chunksize(length.data);

    csr = new OutlierHandler<Data>(length.data, &sp.workspace_nbyte);
}

COMPR_TYPE
void COMPRESSOR::lorenzo_dryrun(Capsule<Data>* in_data)
{
    auto get_npart = [](auto size, auto subsize) { return (size + subsize - 1) / subsize; };

    if (ctx->task_is.dryrun) {
        auto len    = length.data;
        auto eb     = config.eb;
        auto ebx2_r = config.ebx2_r;
        auto ebx2   = config.ebx2;

        logging(log_info, "invoke dry-run");
        constexpr auto SEQ       = 4;
        constexpr auto SUBSIZE   = 256;
        auto           dim_block = SUBSIZE / SEQ;
        auto           dim_grid  = get_npart(len, SUBSIZE);

        cusz::dual_quant_dryrun<Data, float, SUBSIZE, SEQ><<<dim_grid, dim_block>>>(in_data->dptr, len, ebx2_r, ebx2);
        HANDLE_ERROR(hipDeviceSynchronize());

        Data* dryrun_result;
        hipHostMalloc(&dryrun_result, len * sizeof(Data));
        hipMemcpy(dryrun_result, in_data->dptr, len * sizeof(Data), hipMemcpyDeviceToHost);

        analysis::verify_data<Data>(&ctx->stat, dryrun_result, in_data->hptr, len);
        analysis::print_data_quality_metrics<Data>(&ctx->stat, false, eb, 0);

        hipHostFree(dryrun_result);

        exit(0);
    }
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::predict_quantize(Capsule<Data>* data, dim3 xyz, Capsule<Data>* anchor, Capsule<Quant>* quant)
{
    logging(log_info, "compressing...");
    // TODO "predictor" -> "prediction"
    if (ctx->task_is.predictor == "lorenzo") {
        // TODO class lorenzo
        compress_lorenzo_construct<Data, Quant, float>(
            data->dptr, quant->dptr, xyz, ctx->ndim, config.eb, config.radius, time.lossy);
    }
    else if (ctx->task_is.predictor == "spline3d") {
        if (ctx->ndim != 3) throw std::runtime_error("must be 3D data.");
        // TODO timer
        spline3->predict_quantize();
    }
    else {
        throw std::runtime_error("must be \"lorenzo\" or \"spline3d\"");
    }

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::gather_outlier(Capsule<Data>* in_data)
{
    // can be known on Compressor init
    hipMalloc((void**)&sp.workspace, sp.workspace_nbyte);
    hipHostMalloc((void**)&sp.dump, sp.workspace_nbyte);

    csr->configure(sp.workspace)  //
        .gather_CUDA10(in_data->dptr, sp.dump_nbyte, time.outlier)
        .archive(sp.dump, length.nnz_outlier);

    data_seg.nbyte_raw.at("outlier") = sp.dump_nbyte;

    hipFree(sp.workspace);

    auto fmt_nnz = "(" + std::to_string(length.nnz_outlier / 1.0 / length.data * 100) + "%)";
    logging(log_info, "#outlier = ", length.nnz_outlier, fmt_nnz);

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::get_freq_and_codebook(
    Capsule<Quant>*        quant,
    Capsule<unsigned int>* freq,
    Capsule<Huff>*         book,
    Capsule<uint8_t>*      revbook)
{
    wrapper::get_frequency<Quant>(quant->dptr, length.quant, freq->dptr, length.dict_size, time.hist);

    {  // This is end-to-end time for parbook.
        auto t = new cuda_timer_t;
        t->timer_start();
        lossless::par_get_codebook<Quant, Huff>(length.dict_size, freq->dptr, book->dptr, revbook->dptr);
        time.book = t->timer_end_get_elapsed_time();
        hipDeviceSynchronize();
        delete t;
    }

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::analyze_compressibility(
    Capsule<unsigned int>* freq,  //
    Capsule<Huff>*         book)
{
    if (ctx->report.compressibility) {
        hipHostMalloc(&freq->hptr, freq->nbyte()), freq->d2h();
        hipHostMalloc(&book->hptr, book->nbyte()), book->d2h();

        Analyzer analyzer{};
        analyzer  //
            .EstimateFromHistogram(freq->hptr, length.dict_size)
            .template GetHuffmanCodebookStat<Huff>(freq->hptr, book->hptr, length.data, length.dict_size)
            .PrintCompressibilityInfo(true);

        hipHostFree(freq->hptr);
        hipHostFree(book->hptr);
    }

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::internal_eval_try_export_book(Capsule<Huff>* book)
{
    // internal evaluation, not stored in sz archive
    if (ctx->task_is.export_book) {
        hipHostMalloc(&book->hptr, length.dict_size * sizeof(decltype(book->hptr)));
        book->d2h();

        std::stringstream s;
        s << ctx->fnames.path_basename + "-" << length.dict_size << "-ui" << sizeof(Huff) << ".lean-book";

        // TODO as part of dump
        io::write_array_to_binary(s.str(), book->hptr, length.dict_size);

        hipHostFree(book->hptr);
        book->hptr = nullptr;

        logging(log_info, "exporting codebook as binary; suffix: \".lean-book\"");

        data_seg.nbyte_raw.at("book") = length.dict_size * sizeof(Huff);
    }
    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::internal_eval_try_export_quant(Capsule<Quant>* quant)
{
    // internal_eval
    if (ctx->task_is.export_quant) {  //
        hipHostMalloc(&quant->hptr, quant->nbyte());
        quant->d2h();

        data_seg.nbyte_raw.at("quant") = quant->nbyte();

        // TODO as part of dump
        io::write_array_to_binary(ctx->fnames.path_basename + ".lean-quant", quant->hptr, length.quant);
        logging(log_info, "exporting quant as binary; suffix: \".lean-quant\"");
        logging(log_info, "exiting");
        exit(0);
    }
    return *this;
}

COMPR_TYPE
void COMPRESSOR::try_skip_huffman(Capsule<Quant>* quant)
{
    // decide if skipping Huffman coding
    if (ctx->task_is.skip_huffman) {
        hipHostMalloc(&quant->hptr, quant->nbyte());
        quant->d2h();

        // TODO: as part of cusza
        io::write_array_to_binary(ctx->fnames.path_basename + ".quant", quant->hptr, length.quant);
        logging(log_info, "to store quant.code directly (Huffman enc skipped)");
        exit(0);
    }
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::try_report_time()
{
    if (ctx->report.time)
        report_compression_time(length.data, time.lossy, time.outlier, time.hist, time.book, time.lossless);
    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::export_revbook(Capsule<uint8_t>* revbook)
{
    revbook->d2h();
    data_seg.nbyte_raw.at("revbook") = get_revbook_nbyte();

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::huffman_encode(
    Capsule<Quant>* quant,  //
    Capsule<Huff>*  book)
{
    // fix-length space, padding improvised
    hipMalloc(&huffman.array.d_encspace, sizeof(Huff) * (length.quant + ctx->huffman_chunk + HuffConfig::Db_encode));

    auto nchunk = (length.quant + ctx->huffman_chunk - 1) / ctx->huffman_chunk;
    ctx->nchunk = nchunk;

    // gather metadata (without write) before gathering huff as sp on GPU
    hipHostMalloc(&huffman.array.h_counts, nchunk * 3 * sizeof(size_t));
    hipMalloc(&huffman.array.d_counts, nchunk * 3 * sizeof(size_t));

    auto dev_bits    = huffman.array.d_counts;
    auto dev_uints   = huffman.array.d_counts + nchunk;
    auto dev_entries = huffman.array.d_counts + nchunk * 2;

    lossless::HuffmanEncode<Quant, Huff, false>(
        huffman.array.d_encspace, dev_bits, dev_uints, dev_entries, huffman.array.h_counts,
        //
        nullptr,
        //
        quant->dptr, book->dptr, length.quant, ctx->huffman_chunk, ctx->dict_size, &huffman.meta.num_bits,
        &huffman.meta.num_uints, time.lossless);

    // --------------------------------------------------------------------------------
    hipHostMalloc(&huffman.array.h_bitstream, huffman.meta.num_uints * sizeof(Huff));
    hipMalloc(&huffman.array.d_bitstream, huffman.meta.num_uints * sizeof(Huff));

    lossless::HuffmanEncode<Quant, Huff, true>(
        huffman.array.d_encspace, nullptr, dev_uints, dev_entries, nullptr,
        //
        huffman.array.d_bitstream,
        //
        nullptr, nullptr, length.quant, ctx->huffman_chunk, 0, nullptr, nullptr, time.lossless);

    // --------------------------------------------------------------------------------
    hipMemcpy(
        huffman.array.h_bitstream, huffman.array.d_bitstream, huffman.meta.num_uints * sizeof(Huff),
        hipMemcpyDeviceToHost);

    // TODO size_t -> MetadataT
    data_seg.nbyte_raw.at("huff-meta")      = sizeof(size_t) * (2 * nchunk);
    data_seg.nbyte_raw.at("huff-bitstream") = sizeof(Huff) * huffman.meta.num_uints;

    hipFree(huffman.array.d_encspace);

    huffman.meta.revbook_nbyte = get_revbook_nbyte();

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::pack_metadata()
{
    header->x    = ctx->x;
    header->y    = ctx->y;
    header->z    = ctx->z;
    header->w    = ctx->w;
    header->ndim = ctx->ndim;
    header->eb   = ctx->eb;

    header->outlier.nnz        = length.nnz_outlier;
    header->data_len           = ctx->data_len;
    header->config.quant_nbyte = ctx->quant_nbyte;
    header->config.huff_nbyte  = ctx->huff_nbyte;
    header->huffman.chunk      = ctx->huffman_chunk;
    header->skip_huffman       = ctx->task_is.skip_huffman;

    // header->outlier.num_bits  = huffman.meta.num_bits;
    header->huffman.num_uints = huffman.meta.num_uints;

    header->nbyte.revbook = huffman.meta.revbook_nbyte;

    return *this;
}

////////////////////////////////////////////////////////////////////////////////

#define DECOMPR_TYPE template <typename Data, typename Quant, typename Huff, typename FP>
#define DECOMPRESSOR Decompressor<Data, Quant, Huff, FP>

DECOMPR_TYPE
void DECOMPRESSOR::unpack_metadata()
{
    ctx->x    = header->x;
    ctx->y    = header->y;
    ctx->z    = header->z;
    ctx->w    = header->w;
    ctx->ndim = header->ndim;
    ctx->eb   = header->eb;

    ctx->data_len = header->data_len;

    ctx->quant_nbyte          = header->config.quant_nbyte;
    ctx->huff_nbyte           = header->config.huff_nbyte;
    ctx->huffman_chunk        = header->huffman.chunk;
    ctx->task_is.skip_huffman = header->skip_huffman;

    //
    length.nnz_outlier         = header->outlier.nnz;
    huffman.meta.num_uints     = header->huffman.num_uints;
    huffman.meta.revbook_nbyte = header->nbyte.revbook;

    length.data  = ctx->data_len;
    length.quant = length.data;  // TODO if lorenzo

    config.eb     = ctx->eb;
    config.ebx2   = config.eb * 2;
    config.ebx2_r = 1 / (config.eb * 2);
    config.eb_r   = 1 / config.eb;
}

DECOMPR_TYPE
void DECOMPRESSOR::report_decompression_time(size_t len, float lossy, float outlier, float lossless)
{
    auto display_throughput = [](float time, size_t nbyte) {
        auto throughput = nbyte * 1.0 / (1024 * 1024 * 1024) / (time * 1e-3);
        cout << throughput << "GiB/s\n";
    };
    //
    cout << "\nTIME in milliseconds\t================================================================\n";
    float all = lossy + outlier + lossless;

    printf("TIME\tscatter outlier:\t%f\t", outlier), display_throughput(outlier, len * sizeof(Data));
    printf("TIME\tHuffman decode:\t%f\t", lossless), display_throughput(lossless, len * sizeof(Data));
    printf("TIME\treconstruct:\t%f\t", lossy), display_throughput(lossy, len * sizeof(Data));

    cout << "TIME\t--------------------------------------------------------------------------------\n";

    printf("TIME\tdecompress (sum):\t%f\t", all), display_throughput(all, len * sizeof(Data));

    cout << "TIME\t================================================================================\n\n";
}

DECOMPR_TYPE
DECOMPRESSOR::Decompressor(cusz_header* _header, argpack* _ctx) : header(_header), ctx(_ctx)
{
    logging(log_info, "decompressing...");

    unpack_metadata();

    m   = static_cast<size_t>(ceil(sqrt(length.data)));
    mxm = m * m;

    // TODO is ctx still needed?
    xyz = dim3(header->x, header->y, header->z);

    csr     = new OutlierHandler<Data>(length.data, length.nnz_outlier);
    spline3 = new Spline3<Data*, Quant*, float>();
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::huffman_decode(Capsule<Quant>* quant)
{
    if (ctx->task_is.skip_huffman) {
        // logging(log_info, "load quant.code from filesystem");
        io::read_binary_to_array(ctx->fnames.path_basename + ".quant", quant->hptr, quant->len);
        quant->h2d();
    }
    else {
        // logging(log_info, "Huffman decode -> quant.code");

        auto basename      = ctx->fnames.path2file;
        auto nchunk        = (ctx->data_len - 1) / ctx->huffman_chunk + 1;
        auto num_uints     = header->huffman.num_uints;
        auto revbook_nbyte = data_seg.nbyte_raw.at("revbook");

        auto host_revbook =
            reinterpret_cast<BYTE*>(consolidated_dump.whole + offsets.at(data_seg.name2order.at("revbook")));

        auto host_in_bitstream =
            reinterpret_cast<Huff*>(consolidated_dump.whole + offsets.at(data_seg.name2order.at("huff-bitstream")));

        auto host_bits_entries =
            reinterpret_cast<size_t*>(consolidated_dump.whole + offsets.at(data_seg.name2order.at("huff-meta")));

        auto dev_out_bitstream = mem::create_devspace_memcpy_h2d(host_in_bitstream, num_uints);
        auto dev_bits_entries  = mem::create_devspace_memcpy_h2d(host_bits_entries, 2 * nchunk);
        auto dev_revbook       = mem::create_devspace_memcpy_h2d(host_revbook, revbook_nbyte);

        lossless::HuffmanDecode<Quant, Huff>(
            dev_out_bitstream, dev_bits_entries, dev_revbook,
            //
            quant, ctx->data_len, ctx->huffman_chunk, huffman.meta.num_uints, ctx->dict_size, time.lossless);

        hipFree(dev_out_bitstream);
        hipFree(dev_bits_entries);
        hipFree(dev_revbook);
    }
    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::scatter_outlier(Data* outlier)
{
    csr_file.host = reinterpret_cast<BYTE*>(consolidated_dump.whole + offsets.at(data_seg.name2order.at("outlier")));
    hipMalloc((void**)&csr_file.dev, csr->bytelen.total);

    hipMemcpy(csr_file.dev, csr_file.host, csr->bytelen.total, hipMemcpyHostToDevice);

    csr->extract(csr_file.dev).scatter_CUDA10(outlier, time.outlier);

    hipFree(csr_file.dev);

    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::reversed_predict_quantize(Data* xdata, dim3 xyz, Data* anchor, Quant* quant)
{
    if (ctx->task_is.predictor == "lorenzo") {
        // TODO lorenzo class
        decompress_lorenzo_reconstruct<Data, Quant, FP>(
            xdata, quant, xyz, ctx->ndim, config.eb, ctx->radius, time.lossy);
    }
    else if (ctx->task_is.predictor == "spline3d") {
        throw std::runtime_error("spline not impl'ed");
        if (ctx->ndim != 3) throw std::runtime_error("Spline3D must be for 3D data.");
        // TODO
        spline3->reversed_predict_quantize();
    }
    else {
        throw std::runtime_error("need to specify predcitor");
    }

    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::calculate_archive_nbyte()
{
    auto demangle = [](const char* name) -> string {
        int   status = -4;
        char* res    = abi::__cxa_demangle(name, nullptr, nullptr, &status);

        const char* const demangled_name = (status == 0) ? res : name;
        string            ret_val(demangled_name);
        free(res);
        return ret_val;
    };

    if (not ctx->task_is.skip_huffman)
        archive_bytes += huffman.meta.num_uints * sizeof(Huff)  // Huffman coded
                         + huffman.meta.revbook_nbyte;          // chunking metadata and reverse codebook
    else
        archive_bytes += length.quant * sizeof(Quant);
    archive_bytes += length.nnz_outlier * (sizeof(Data) + sizeof(int)) + (m + 1) * sizeof(int);

    if (ctx->task_is.skip_huffman) {
        logging(
            log_info, "dtype is \"", demangle(typeid(Data).name()), "\", and quant. code type is \"",
            demangle(typeid(Quant).name()), "\"; a CR of no greater than ", (sizeof(Data) / sizeof(Quant)),
            " is expected when Huffman codec is skipped.");
    }

    if (ctx->task_is.pre_binning) logging(log_info, "Because of 2x2->1 binning, extra 4x CR is added.");

    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::try_report_time()
{
    if (ctx->report.time) report_decompression_time(length.data, time.lossy, time.outlier, time.lossless);

    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::try_compare(Data* xdata)
{
    // TODO move CR out of verify_data
    if (not ctx->fnames.origin_cmp.empty() and ctx->report.quality) {
        logging(log_info, "compare to the original");

        auto odata = io::read_binary_to_new_array<Data>(ctx->fnames.origin_cmp, length.data);

        analysis::verify_data(&ctx->stat, xdata, odata, length.data);
        analysis::print_data_quality_metrics<Data>(
            &ctx->stat, false, ctx->eb, archive_bytes, ctx->task_is.pre_binning ? 4 : 1, true);

        delete[] odata;
    }
    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::try_write2disk(Data* host_xdata)
{
    logging(log_info, "output:", ctx->fnames.path_basename + ".cuszx");

    if (ctx->task_is.skip_write2disk)
        logging(log_dbg, "skip writing unzipped to filesystem");
    else {
        io::write_array_to_binary(ctx->fnames.path_basename + "cuszx", host_xdata, ctx->data_len);
    }

    return *this;
}

////////////////////////////////////////////////////////////////////////////////

#define DATATYPE Capsule<typename DataTrait<If_FP, DataByte>::Data>

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz_compress(argpack* ctx, DATATYPE* in_data)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    auto xyz = dim3(ctx->x, ctx->y, ctx->z);

    Compressor<Data, Quant, Huff, float> cuszc(ctx);
    cuszc.header = new cusz_header();

    // TODO lorenzo class::get_len_quant
    auto lorenzo_get_len_quant = [&]() -> unsigned int { return ctx->data_len + HuffConfig::Db_encode; };

    unsigned int len_quant = ctx->task_is.predictor == "spline3"  //
                                 ? 1
                                 : lorenzo_get_len_quant();

    cuszc.lorenzo_dryrun(in_data);  // subject to change

    Capsule<Quant> quant(len_quant);
    hipMalloc(&quant.dptr, quant.nbyte());

    Capsule<Data>* anchor = nullptr;
    if (ctx->task_is.predictor == "spline3") {
        // TODO
    }

    Capsule<unsigned int> freq(ctx->dict_size);
    hipMalloc(&freq.dptr, freq.nbyte());

    Capsule<Huff> book(ctx->dict_size);
    hipMalloc(&book.dptr, book.nbyte()), book.memset(0xff);

    Capsule<uint8_t> revbook(cuszc.get_revbook_nbyte());
    hipMalloc(&revbook.dptr, revbook.nbyte());
    hipHostMalloc(&revbook.hptr, revbook.nbyte());  // to write to disk later

    cuszc.huffman.array.h_revbook = revbook.hptr;

    cuszc  //
        .predict_quantize(in_data, xyz, anchor, &quant)
        .gather_outlier(in_data)
        .try_skip_huffman(&quant);

    // release in_data; subject to change
    hipFree(in_data->dptr);

    cuszc.get_freq_and_codebook(&quant, &freq, &book, &revbook)
        .analyze_compressibility(&freq, &book)
        .internal_eval_try_export_book(&book)
        .internal_eval_try_export_quant(&quant)
        .export_revbook(&revbook)
        .huffman_encode(&quant, &book)
        .try_report_time()
        .pack_metadata()
        .consolidate();

    hipFree(quant.dptr), hipFree(freq.dptr), hipFree(book.dptr), hipFree(revbook.dptr);
    hipHostFree(revbook.hptr);
    delete cuszc.header;
}

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz_decompress(argpack* ctx)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    auto __cusz_get_filesize = [](std::string fname) -> size_t {
        std::ifstream in(fname.c_str(), std::ifstream::ate | std::ifstream::binary);
        return in.tellg();
    };

    auto fname_dump = ctx->fnames.path2file + ".cusza";
    auto dump_nbyte = __cusz_get_filesize(fname_dump);
    auto h_dump     = io::read_binary_to_new_array<BYTE>(fname_dump, dump_nbyte);

    // cout << "dump-nbyte by tellg()\t" << dump_nbyte << '\n';
    auto header = reinterpret_cast<cusz_header*>(h_dump);

    // TODO Decompressor encapsulate dump
    // TODO float -> another parameter FP
    Decompressor<Data, Quant, Huff, float> cuszd(header, ctx);

    cuszd.consolidated_dump.whole = h_dump;

    cuszd.read_array_nbyte_from_header();
    cuszd.get_data_seg_offsets();

    // TODO lorenzo class::get_len_quant
    auto lorenzo_get_len_quant = [&]() -> unsigned int { return ctx->data_len; };

    unsigned int len_quant = ctx->task_is.predictor == "spline3"  //
                                 ? cuszd.spline3->get_len_quant()
                                 : lorenzo_get_len_quant();

    Capsule<Data>* anchor =
        new Capsule<Data>(cuszd.spline3->get_len_anchor());  // TODO this .dptr is nullable, error-prone
    if (ctx->task_is.predictor == "spline3") {
        hipMalloc(&anchor->dptr, anchor->nbyte());
        hipHostMalloc(&anchor->hptr, anchor->nbyte());

        // TODO dummy, need source
    }

    Capsule<Quant> quant(len_quant);
    hipMalloc(&quant.dptr, quant.nbyte());
    hipHostMalloc(&quant.hptr, quant.nbyte());

    // TODO cuszd.get_len_data_space()
    Capsule<Data> decomp_space(cuszd.mxm + MetadataTrait<1>::Block);  // TODO ad hoc size
    hipMalloc(&decomp_space.dptr, decomp_space.nbyte());
    hipHostMalloc(&decomp_space.hptr, decomp_space.nbyte());
    auto xdata = decomp_space.dptr, outlier = decomp_space.dptr;

    cuszd.huffman_decode(&quant)
        .scatter_outlier(outlier)
        .reversed_predict_quantize(xdata, cuszd.xyz, anchor->dptr, quant.dptr)
        .try_report_time();

    // copy decompressed data to host
    decomp_space.d2h();

    cuszd
        .calculate_archive_nbyte()  //
        .try_compare(decomp_space.hptr)
        .try_write2disk(decomp_space.hptr);
}

////////////////////////////////////////////////////////////////////////////////

template class Compressor<float, uint8_t, uint32_t, float>;
template class Compressor<float, uint16_t, uint32_t, float>;
template class Compressor<float, uint32_t, uint32_t, float>;
template class Compressor<float, uint8_t, unsigned long long, float>;
template class Compressor<float, uint16_t, unsigned long long, float>;
template class Compressor<float, uint32_t, unsigned long long, float>;

template class Decompressor<float, uint8_t, uint32_t, float>;
template class Decompressor<float, uint16_t, uint32_t, float>;
template class Decompressor<float, uint32_t, uint32_t, float>;
template class Decompressor<float, uint8_t, unsigned long long, float>;
template class Decompressor<float, uint16_t, unsigned long long, float>;
template class Decompressor<float, uint32_t, unsigned long long, float>;

#define CUSZ_COMPRESS(DBYTE, QBYTE, HBYTE) \
    template void cusz_compress<true, DBYTE, QBYTE, HBYTE>(argpack*, Capsule<float>*);

CUSZ_COMPRESS(4, 1, 4)
CUSZ_COMPRESS(4, 1, 8)
CUSZ_COMPRESS(4, 2, 4)
CUSZ_COMPRESS(4, 2, 8)

#define CUSZ_DECOMPRESS(DBYTE, QBYTE, HBYTE) template void cusz_decompress<true, DBYTE, QBYTE, HBYTE>(argpack*);

CUSZ_DECOMPRESS(4, 1, 4)
CUSZ_DECOMPRESS(4, 1, 8)
CUSZ_DECOMPRESS(4, 2, 4)
CUSZ_DECOMPRESS(4, 2, 8)
