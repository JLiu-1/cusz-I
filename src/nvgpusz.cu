#include "hip/hip_runtime.h"
/**
 * @file nvgpusz.cu
 * @author Jiannan Tian
 * @brief Workflow of cuSZ.
 * @version 0.3
 * @date 2021-07-12
 * (create) 2020-02-12; (release) 2020-09-20; (rev.1) 2021-01-16; (rev.2) 2021-07-12; (rev.3) 2021-09-06
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cxxabi.h>
#include <bitset>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <type_traits>
#include <typeinfo>

#include "analysis/analyzer.hh"
#include "argparse.hh"
#include "kernel/dryrun.cuh"
#include "kernel/lorenzo.cuh"
#include "metadata.hh"
#include "nvgpusz.cuh"
#include "type_trait.hh"
#include "utils.hh"
#include "wrapper/extrap_lorenzo.cuh"
#include "wrapper/handle_sparsity.cuh"
#include "wrapper/huffman_enc_dec.cuh"
#include "wrapper/huffman_parbook.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;

////////////////////////////////////////////////////////////////////////////////

#define COMPR_TYPE template <typename Data, typename Quant, typename Huff, typename FP>
#define COMPRESSOR Compressor<Data, Quant, Huff, FP>

COMPR_TYPE
unsigned int COMPRESSOR::tune_deflate_chunksize(size_t len)
{
    cout << "autotuning deflate chunksize\n";

    int current_dev = 0;
    hipSetDevice(current_dev);
    hipDeviceProp_t dev_prop{};
    hipGetDeviceProperties(&dev_prop, current_dev);

    auto nSM                = dev_prop.multiProcessorCount;
    auto allowed_block_dim  = dev_prop.maxThreadsPerBlock;
    auto deflate_nthread    = allowed_block_dim * nSM / HuffConfig::deflate_constant;
    auto optimal_chunk_size = (len + deflate_nthread - 1) / deflate_nthread;
    optimal_chunk_size      = ((optimal_chunk_size - 1) / HuffConfig::Db_deflate + 1) * HuffConfig::Db_deflate;

    return optimal_chunk_size;
}

COMPR_TYPE
void COMPRESSOR::report_compression_time(size_t len, float lossy, float outlier, float hist, float book, float lossless)
{
    auto display_throughput = [](float time, size_t nbyte) {
        auto throughput = nbyte * 1.0 / (1024 * 1024 * 1024) / (time * 1e-3);
        cout << throughput << "GiB/s\n";
    };
    //
    cout << "\nTIME in milliseconds\t================================================================\n";
    float nonbook = lossy + outlier + hist + lossless;

    printf("TIME\tconstruct:\t%f\t", lossy), display_throughput(lossy, len * sizeof(Data));
    printf("TIME\toutlier:\t%f\t", outlier), display_throughput(outlier, len * sizeof(Data));
    printf("TIME\thistogram:\t%f\t", hist), display_throughput(hist, len * sizeof(Data));
    printf("TIME\tencode:\t%f\t", lossless), display_throughput(lossless, len * sizeof(Data));

    cout << "TIME\t--------------------------------------------------------------------------------\n";
    printf("TIME\tnon-book kernels (sum):\t%f\t", nonbook), display_throughput(nonbook, len * sizeof(Data));
    cout << "TIME\t================================================================================\n";
    printf("TIME\tbuild book (not counted in prev section):\t%f\t", book), display_throughput(book, len * sizeof(Data));
    printf("TIME\t*all* kernels (sum, count book time):\t%f\t", nonbook + book),
        display_throughput(nonbook + book, len * sizeof(Data));
    cout << "TIME\t================================================================================\n\n";
}

COMPR_TYPE
void COMPRESSOR::export_codebook(Huff* d_book, const string& basename, size_t dict_size)
{
    auto              h_book = mem::create_devspace_memcpy_d2h(d_book, dict_size);
    std::stringstream s;
    s << basename + "-" << dict_size << "-ui" << sizeof(Huff) << ".lean-book";
    logging(log_dbg, "export \"lean\" codebook (of dict_size) as", s.str());
    io::write_array_to_binary(s.str(), h_book, dict_size);
    hipHostFree(h_book);
    h_book = nullptr;
}

COMPR_TYPE
COMPRESSOR::Compressor(argpack* _ap, unsigned int _data_len, double _eb)
{
    ap = _ap;

    ndim = ap->ndim;

    config.radius = ap->radius;

    length.data      = _data_len;
    length.quant     = length.data;  // TODO if lorenzo
    length.dict_size = ap->dict_size;

    config.eb     = _eb;
    config.ebx2   = _eb * 2;
    config.ebx2_r = 1 / (_eb * 2);
    config.eb_r   = 1 / _eb;

    if (ap->sz_workflow.autotune_huffchunk) ap->huffman_chunk = tune_deflate_chunksize(length.data);
}

COMPR_TYPE
void COMPRESSOR::lorenzo_dryrun(struct PartialData<Data>* in_data)
{
    auto get_npart = [](auto size, auto subsize) { return (size + subsize - 1) / subsize; };

    if (ap->sz_workflow.dryrun) {
        auto len    = length.data;
        auto eb     = config.eb;
        auto ebx2_r = config.ebx2_r;
        auto ebx2   = config.ebx2;

        logging(log_info, "invoke dry-run");
        constexpr auto SEQ       = 4;
        constexpr auto SUBSIZE   = 256;
        auto           dim_block = SUBSIZE / SEQ;
        auto           dim_grid  = get_npart(len, SUBSIZE);

        cusz::dual_quant_dryrun<Data, float, SUBSIZE, SEQ><<<dim_grid, dim_block>>>(in_data->dptr, len, ebx2_r, ebx2);
        HANDLE_ERROR(hipDeviceSynchronize());

        Data* dryrun_result;
        hipHostMalloc(&dryrun_result, len * sizeof(Data));
        hipMemcpy(dryrun_result, in_data->dptr, len * sizeof(Data), hipMemcpyDeviceToHost);

        analysis::verify_data<Data>(&ap->stat, dryrun_result, in_data->hptr, len);
        analysis::print_data_quality_metrics<Data>(&ap->stat, false, eb, 0);

        hipHostFree(dryrun_result);

        exit(0);
    }
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::predict_quantize(struct PartialData<Data>* data, dim3 xyz, struct PartialData<Quant>* quant)
{
    logging(log_info, "invoke lossy-construction");
    if (ap->sz_workflow.predictor == "lorenzo") {
        compress_lorenzo_construct<Data, Quant, float>(
            data->dptr, quant->dptr, xyz, ap->ndim, config.eb, config.radius, time.lossy);
    }
    else if (ap->sz_workflow.predictor == "spline3d") {
        throw std::runtime_error("spline not impl'ed");
        if (ap->ndim != 3) throw std::runtime_error("Spline3D must be for 3D data.");
        // compress_spline3d_construct<Data, Quant, float>(
        //     in_data->dptr, quant.dptr, xyz, ap->ndim, eb, radius, time_lossy);
    }
    else {
        throw std::runtime_error("need to specify predcitor");
    }

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::gather_outlier(struct PartialData<Data>* in_data)
{
    unsigned int workspace_nbyte, dump_nbyte;
    uint8_t *    workspace, *dump;
    workspace_nbyte = get_compress_sparse_workspace<Data>(length.data);
    hipMalloc((void**)&workspace, workspace_nbyte);
    hipHostMalloc((void**)&dump, workspace_nbyte);

    OutlierHandler<Data> csr(length.data);
    csr.configure(workspace)  //
        .gather_CUDA10(in_data->dptr, dump_nbyte, time.outlier)
        .archive(dump, length.nnz_outlier);
    io::write_array_to_binary(ap->subfiles.compress.out_outlier, dump, dump_nbyte);

    hipFree(workspace), hipHostFree(dump);

    auto fmt_nnz = "(" + std::to_string(length.nnz_outlier / 1.0 / length.data * 100) + "%)";
    logging(log_info, "nnz/#outlier:", length.nnz_outlier, fmt_nnz, "saved");

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::get_freq_and_codebook(
    struct PartialData<Quant>*        quant,
    struct PartialData<unsigned int>* freq,
    struct PartialData<Huff>*         book,
    struct PartialData<uint8_t>*      revbook)
{
    wrapper::get_frequency<Quant>(quant->dptr, length.quant, freq->dptr, length.dict_size, time.hist);

    {  // This is end-to-end time for parbook.
        auto t = new cuda_timer_t;
        t->timer_start();
        lossless::par_get_codebook<Quant, Huff>(length.dict_size, freq->dptr, book->dptr, revbook->dptr);
        time.book = t->timer_end_get_elapsed_time();
        hipDeviceSynchronize();
        delete t;
    }

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::analyze_compressibility(
    struct PartialData<unsigned int>* freq,  //
    struct PartialData<Huff>*         book)
{
    if (ap->report.compressibility) {
        hipHostMalloc(&freq->hptr, freq->nbyte()), freq->d2h();
        hipHostMalloc(&book->hptr, book->nbyte()), book->d2h();

        Analyzer analyzer{};
        analyzer  //
            .EstimateFromHistogram(freq->hptr, length.dict_size)
            .template GetHuffmanCodebookStat<Huff>(freq->hptr, book->hptr, length.data, length.dict_size)
            .PrintCompressibilityInfo(true);

        hipHostFree(freq->hptr);
        hipHostFree(book->hptr);
    }

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::internal_eval_try_export_book(struct PartialData<Huff>* book)
{
    // internal evaluation, not stored in sz archive
    if (ap->sz_workflow.export_book) {  //
        export_codebook(book->dptr, ap->subfiles.compress.huff_base, length.dict_size);
        logging(log_info, "exporting codebook as binary; suffix: \".lean-book\"");
    }
    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::internal_eval_try_export_quant(struct PartialData<Quant>* quant)
{
    // internal_eval
    if (ap->sz_workflow.export_quant) {  //
        hipHostMalloc(&quant->hptr, quant->nbyte());
        quant->d2h();

        io::write_array_to_binary(ap->subfiles.compress.raw_quant, quant->hptr, length.quant);
        logging(log_info, "exporting quant as binary; suffix: \".lean-quant\"");
        logging(log_info, "exiting");
        exit(0);
    }
    return *this;
}

COMPR_TYPE
void COMPRESSOR::try_skip_huffman(struct PartialData<Quant>* quant)
{
    // decide if skipping Huffman coding
    if (ap->sz_workflow.skip_huffman) {
        hipHostMalloc(&quant->hptr, quant->nbyte());
        quant->d2h();

        io::write_array_to_binary(ap->subfiles.compress.out_quant, quant->hptr, length.quant);
        logging(log_info, "to store quant.code directly (Huffman enc skipped)");
        exit(0);
    }
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::try_report_time()
{
    if (ap->report.time)
        report_compression_time(length.data, time.lossy, time.outlier, time.hist, time.book, time.lossless);
    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::export_revbook(struct PartialData<uint8_t>* revbook)
{
    revbook->d2h();
    io::write_array_to_binary(ap->subfiles.compress.huff_base + ".canon", revbook->hptr, get_revbook_nbyte());
    hipHostFree(revbook->hptr);

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::huffman_encode(
    struct PartialData<Quant>* quant,  //
    struct PartialData<Huff>*  book)
{
    lossless::HuffmanEncode<Quant, Huff>(
        ap->subfiles.compress.huff_base, quant->dptr, book->dptr, length.quant, ap->huffman_chunk, ap->dict_size,
        huffman_meta.num_bits, huffman_meta.num_uints, time.lossless);

    huffman_meta.revbook_nbyte = get_revbook_nbyte();

    return *this;
}

COMPR_TYPE
COMPRESSOR& COMPRESSOR::pack_metadata(metadata_pack* mp)
{
    mp->dim4    = ap->dim4;
    mp->stride4 = ap->stride4;
    mp->nblk4   = ap->nblk4;
    mp->ndim    = ap->ndim;
    mp->eb      = ap->eb;
    mp->len     = ap->len;

    mp->nnz = length.nnz_outlier;

    if (ap->dtype == "f32") mp->dtype = DataType::kF32;
    if (ap->dtype == "f64") mp->dtype = DataType::kF64;

    mp->quant_byte    = ap->quant_byte;
    mp->huff_byte     = ap->huff_byte;
    mp->huffman_chunk = ap->huffman_chunk;
    mp->skip_huffman  = ap->sz_workflow.skip_huffman;

    mp->num_bits      = huffman_meta.num_bits;
    mp->num_uints     = huffman_meta.num_uints;
    mp->revbook_nbyte = huffman_meta.revbook_nbyte;

    return *this;
}

////////////////////////////////////////////////////////////////////////////////

#define DECOMPR_TYPE template <typename Data, typename Quant, typename Huff, typename FP>
#define DECOMPRESSOR Decompressor<Data, Quant, Huff, FP>

DECOMPR_TYPE
void DECOMPRESSOR::unpack_metadata(metadata_pack* mp, argpack* ap)
{
    ap->dim4    = mp->dim4;
    ap->stride4 = mp->stride4;
    ap->nblk4   = mp->nblk4;
    ap->ndim    = mp->ndim;
    ap->eb      = mp->eb;
    ap->len     = mp->len;

    if (mp->dtype == DataType::kF32) ap->dtype = "f32";
    if (mp->dtype == DataType::kF64) ap->dtype = "f64";

    ap->quant_byte               = mp->quant_byte;
    ap->huff_byte                = mp->huff_byte;
    ap->huffman_chunk            = mp->huffman_chunk;
    ap->sz_workflow.skip_huffman = mp->skip_huffman;
}

DECOMPR_TYPE
void DECOMPRESSOR::report_decompression_time(size_t len, float lossy, float outlier, float lossless)
{
    auto display_throughput = [](float time, size_t nbyte) {
        auto throughput = nbyte * 1.0 / (1024 * 1024 * 1024) / (time * 1e-3);
        cout << throughput << "GiB/s\n";
    };
    //
    cout << "\nTIME in milliseconds\t================================================================\n";
    float all = lossy + outlier + lossless;

    printf("TIME\tscatter outlier:\t%f\t", outlier), display_throughput(outlier, len * sizeof(Data));
    printf("TIME\tHuffman decode:\t%f\t", lossless), display_throughput(lossless, len * sizeof(Data));
    printf("TIME\treconstruct:\t%f\t", lossy), display_throughput(lossy, len * sizeof(Data));

    cout << "TIME\t--------------------------------------------------------------------------------\n";

    printf("TIME\tdecompress (sum):\t%f\t", all), display_throughput(all, len * sizeof(Data));

    cout << "TIME\t================================================================================\n\n";
}

DECOMPR_TYPE
DECOMPRESSOR::Decompressor(metadata_pack* _mp, argpack* _ap)
{
    logging(log_info, "invoke lossy-reconstruction");

    unpack_metadata(_mp, _ap);

    length.nnz_outlier         = _mp->nnz;
    huffman_meta.num_uints     = _mp->num_uints;
    huffman_meta.revbook_nbyte = _mp->revbook_nbyte;

    ap           = _ap;
    length.data  = ap->len;
    length.quant = length.data;  // TODO if lorenzo

    config.eb     = ap->eb;
    config.ebx2   = config.eb * 2;
    config.ebx2_r = 1 / (config.eb * 2);
    config.eb_r   = 1 / config.eb;

    m   = static_cast<size_t>(ceil(sqrt(length.data)));
    mxm = m * m;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::huffman_decode(struct PartialData<Quant>* quant)
{
    if (ap->sz_workflow.skip_huffman) {
        logging(log_info, "load quant.code from filesystem");
        io::read_binary_to_array(ap->subfiles.decompress.in_quant, quant->hptr, quant->len);
        quant->h2d();
    }
    else {
        logging(log_info, "Huffman decode -> quant.code");
        lossless::HuffmanDecode<Quant, Huff>(
            ap->subfiles.path2file, quant, ap->len, ap->huffman_chunk, huffman_meta.num_uints, ap->dict_size,
            time.lossless);
    }
    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::scatter_outlier(Data* outlier)
{
    OutlierHandler<Data> csr(length.data, length.nnz_outlier);

    uint8_t *h_csr_file, *d_csr_file;
    hipHostMalloc((void**)&h_csr_file, csr.bytelen.total);
    hipMalloc((void**)&d_csr_file, csr.bytelen.total);

    io::read_binary_to_array<uint8_t>(ap->subfiles.decompress.in_outlier, h_csr_file, csr.bytelen.total);
    hipMemcpy(d_csr_file, h_csr_file, csr.bytelen.total, hipMemcpyHostToDevice);

    csr.extract(d_csr_file).scatter_CUDA10(outlier, time.outlier);

    hipHostFree(h_csr_file);
    hipFree(d_csr_file);

    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::reversed_predict_quantize(Data* xdata, Quant* quant, dim3 xyz)
{
    if (ap->sz_workflow.predictor == "lorenzo") {
        decompress_lorenzo_reconstruct<Data, Quant, FP>(xdata, quant, xyz, ap->ndim, config.eb, ap->radius, time.lossy);
    }
    else if (ap->sz_workflow.predictor == "spline3d") {
        throw std::runtime_error("spline not impl'ed");
        if (ap->ndim != 3) throw std::runtime_error("Spline3D must be for 3D data.");
        // decompress_spline3d_reconstruct(xdata, quant.dptr, xyz, ap->ndim, eb, radius, time_lossy);
    }
    else {
        throw std::runtime_error("need to specify predcitor");
    }

    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::calculate_archive_nbyte()
{
    auto demangle = [](const char* name) -> string {
        int   status = -4;
        char* res    = abi::__cxa_demangle(name, nullptr, nullptr, &status);

        const char* const demangled_name = (status == 0) ? res : name;
        string            ret_val(demangled_name);
        free(res);
        return ret_val;
    };

    if (not ap->sz_workflow.skip_huffman)
        archive_bytes += huffman_meta.num_uints * sizeof(Huff)  // Huffman coded
                         + huffman_meta.revbook_nbyte;          // chunking metadata and reverse codebook
    else
        archive_bytes += length.quant * sizeof(Quant);
    archive_bytes += length.nnz_outlier * (sizeof(Data) + sizeof(int)) + (m + 1) * sizeof(int);

    if (ap->sz_workflow.skip_huffman) {
        logging(
            log_info, "dtype is \"", demangle(typeid(Data).name()), "\", and quant. code type is \"",
            demangle(typeid(Quant).name()), "\"; a CR of no greater than ", (sizeof(Data) / sizeof(Quant)),
            " is expected when Huffman codec is skipped.");
    }

    if (ap->sz_workflow.pre_binning) logging(log_info, "Because of 2x2->1 binning, extra 4x CR is added.");

    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::try_report_time()
{
    if (ap->report.time) report_decompression_time(length.data, time.lossy, time.outlier, time.lossless);

    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::try_compare(Data* xdata)
{
    // TODO move CR out of verify_data
    if (not ap->subfiles.decompress.in_origin.empty() and ap->report.quality) {
        logging(log_info, "load the original datum for comparison");

        auto odata = io::read_binary_to_new_array<Data>(ap->subfiles.decompress.in_origin, length.data);

        analysis::verify_data(&ap->stat, xdata, odata, length.data);
        analysis::print_data_quality_metrics<Data>(
            &ap->stat, false, ap->eb, archive_bytes, ap->sz_workflow.pre_binning ? 4 : 1, true);

        delete[] odata;
    }
    return *this;
}

DECOMPR_TYPE
DECOMPRESSOR& DECOMPRESSOR::try_write2disk(Data* host_xdata)
{
    logging(log_info, "output:", ap->subfiles.path2file + ".szx");

    if (ap->sz_workflow.skip_write2disk)
        logging(log_dbg, "skip writing unzipped to filesystem");
    else {
        io::write_array_to_binary(ap->subfiles.decompress.out_xdata, host_xdata, ap->len);
    }

    return *this;
}

////////////////////////////////////////////////////////////////////////////////

#define DATATYPE struct PartialData<typename DataTrait<If_FP, DataByte>::Data>

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz_compress(argpack* ap, DATATYPE* in_data, dim3 xyz, metadata_pack* mp, unsigned int optional_w)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    Compressor<Data, Quant, Huff, float> cuszc(ap, ap->len, ap->eb);

    cuszc.lorenzo_dryrun(in_data);  // subject to change

    struct PartialData<Quant> quant(ap->len + HuffConfig::Db_encode);
    hipMalloc(&quant.dptr, quant.nbyte());

    struct PartialData<unsigned int> freq(ap->dict_size);
    hipMalloc(&freq.dptr, freq.nbyte());

    struct PartialData<Huff> book(ap->dict_size);
    hipMalloc(&book.dptr, book.nbyte()), book.memset(0xff);

    struct PartialData<uint8_t> revbook(cuszc.get_revbook_nbyte());
    hipMalloc(&revbook.dptr, revbook.nbyte());
    hipHostMalloc(&revbook.hptr, revbook.nbyte());  // to write to disk later

    cuszc  //
        .predict_quantize(in_data, xyz, &quant)
        .gather_outlier(in_data)
        .try_skip_huffman(&quant);

    // release in_data; subject to change
    hipFree(in_data->dptr);

    cuszc.get_freq_and_codebook(&quant, &freq, &book, &revbook)
        .analyze_compressibility(&freq, &book)
        .internal_eval_try_export_book(&book)
        .internal_eval_try_export_quant(&quant)
        .export_revbook(&revbook)
        .huffman_encode(&quant, &book)
        .try_report_time()
        .pack_metadata(mp);

    hipFree(quant.dptr), hipFree(freq.dptr), hipFree(book.dptr), hipFree(revbook.dptr);
}

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz_decompress(argpack* ap, metadata_pack* mp)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    Decompressor<Data, Quant, Huff, float> cuszd(mp, ap);

    auto xyz = dim3(ap->dim4._0, ap->dim4._1, ap->dim4._2);

    struct PartialData<Quant> quant(cuszd.length.quant);
    hipMalloc(&quant.dptr, quant.nbyte());
    hipHostMalloc(&quant.hptr, quant.nbyte());

    struct PartialData<Data> _data(cuszd.mxm + MetadataTrait<1>::Block);  // TODO ad hoc size
    hipMalloc(&_data.dptr, _data.nbyte());
    hipHostMalloc(&_data.hptr, _data.nbyte());
    auto xdata = _data.dptr, outlier = _data.dptr;

    cuszd.huffman_decode(&quant)
        .scatter_outlier(outlier)
        .reversed_predict_quantize(xdata, quant.dptr, xyz)
        .try_report_time();

    // copy decompressed data to host
    _data.d2h();

    cuszd
        .calculate_archive_nbyte()  //
        .try_compare(_data.hptr)
        .try_write2disk(_data.hptr);
}

////////////////////////////////////////////////////////////////////////////////

template class Compressor<float, uint8_t, uint32_t, float>;
template class Compressor<float, uint16_t, uint32_t, float>;
template class Compressor<float, uint32_t, uint32_t, float>;
template class Compressor<float, uint8_t, unsigned long long, float>;
template class Compressor<float, uint16_t, unsigned long long, float>;
template class Compressor<float, uint32_t, unsigned long long, float>;

template class Decompressor<float, uint8_t, uint32_t, float>;
template class Decompressor<float, uint16_t, uint32_t, float>;
template class Decompressor<float, uint32_t, uint32_t, float>;
template class Decompressor<float, uint8_t, unsigned long long, float>;
template class Decompressor<float, uint16_t, unsigned long long, float>;
template class Decompressor<float, uint32_t, unsigned long long, float>;

#define CUSZ_COMPRESS(DBYTE, QBYTE, HBYTE)                  \
    template void cusz_compress<true, DBYTE, QBYTE, HBYTE>( \
        argpack*, struct PartialData<float>*, dim3, metadata_pack*, unsigned int);

CUSZ_COMPRESS(4, 1, 4)
CUSZ_COMPRESS(4, 1, 8)
CUSZ_COMPRESS(4, 2, 4)
CUSZ_COMPRESS(4, 2, 8)

#define CUSZ_DECOMPRESS(DBYTE, QBYTE, HBYTE) \
    template void cusz_decompress<true, DBYTE, QBYTE, HBYTE>(argpack*, metadata_pack*);

CUSZ_DECOMPRESS(4, 1, 4)
CUSZ_DECOMPRESS(4, 1, 8)
CUSZ_DECOMPRESS(4, 2, 4)
CUSZ_DECOMPRESS(4, 2, 8)
