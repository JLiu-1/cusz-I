#include "hip/hip_runtime.h"

#include <iostream>
#include "../src/common/type_traits.hh"
#include "../src/kernel/lorenzo.h"
#include "../src/kernel/prototype_lorenzo.cuh"
#include "../src/metadata.hh"
#include "../src/utils/cuda_err.cuh"
using std::cerr;
using std::cout;

using Data  = float;
using Quant = unsigned short;
Data*  data;
Quant* quant;
Data*  outlier;
auto   radius = 0;
auto   ebx2 = 1.0, ebx2_r = 1.0;
auto   unified_size = 512 * 512 * 512;

__global__ void dummy() { float data = threadIdx.x; }

void Test1D(int n = 1)
{
    auto dimx = 512 * 512 * 512;

    static const auto SEQ       = ChunkingTrait<1>::SEQ;
    static const auto SUBSIZE   = ChunkingTrait<1>::BLOCK;
    auto              dim_block = DataSubsize / SEQ;
    auto              dim_grid  = ConfigHelper::get_npart(dimx, SUBSIZE);

    for (auto i = 0; i < n; i++) {
        cout << "1Dc " << i << '\n';
        cusz::c_lorenzo_1d1l<Data, Quant, float, SUBSIZE, SEQ><<<dim_grid, dim_block>>>  //
            (data, quant, dimx, radius, ebx2_r);
        HANDLE_ERROR(hipDeviceSynchronize());
    }

    for (auto i = 0; i < n; i++) {
        cout << "1Dx " << i << '\n';
        cusz::x_lorenzo_1d1l<Data, Quant><<<dim_grid, dim_block>>>  //
            (data, outlier, quant, dimx, radius, ebx2);
        HANDLE_ERROR(hipDeviceSynchronize());
    }
}

void Test2D(int n = 1)
{
    auto dimx = 512 * 32, dimy = 512 * 16;
    auto stridey = dimx;

    auto dim_block = dim3(16, 2);
    auto dim_grid  = dim3(
        num_partitions(dimx, 16),  //
        num_partitions(dimy, 16));

    for (auto i = 0; i < n; i++) {
        cout << "2Dc " << i << '\n';
        cusz::c_lorenzo_2d1l_16x16data_mapto16x2<Data, Quant, float><<<dim_grid, dim_block>>>  //
            (data, quant, dimx, dimy, stridey, radius, ebx2_r);
        HANDLE_ERROR(hipDeviceSynchronize());
    }

    for (auto i = 0; i < n; i++) {
        cout << "2Dx " << i << '\n';
        cusz::x_lorenzo_2d1l_16x16data_mapto16x2<Data, Quant><<<dim_grid, dim_block>>>  //
            (data, outlier, quant, dimx, dimy, stridey, radius, ebx2);
        HANDLE_ERROR(hipDeviceSynchronize());
    }
}

void Test3D(int n = 1)
{
    auto dimx = 512, dimy = 512, dimz = 512;
    auto stridey = 512, stridez = 512 * 512;

    auto dim_block = dim3(32, 1, 8);
    auto dim_grid  = dim3(
        num_partitions(dimx, 32),  //
        num_partitions(dimy, 8),   //
        num_partitions(dimz, 8)    //
    );

    for (auto i = 0; i < n; i++) {
        cout << "3Dc " << i << '\n';
        cusz::c_lorenzo_3d1l_32x8x8data_mapto32x1x8<Data, Quant><<<dim_grid, dim_block>>>  //
            (data, quant, dimx, dimy, dimz, stridey, stridez, radius, ebx2_r);
        HANDLE_ERROR(hipDeviceSynchronize());
    }

    for (auto i = 0; i < n; i++) {
        cout << "3Dx " << i << '\n';
        cusz::x_lorenzo_3d1l_32x8x8data_mapto32x1x8<<<dim_grid, dim_block>>>  //
            (data, outlier, quant, dimx, dimy, dimz, stridey, stridez, radius, ebx2);
        HANDLE_ERROR(hipDeviceSynchronize());
    }
}

int main(int argc, char** argv)
{
    hipMallocManaged(&data, unified_size * sizeof(Data));
    hipMallocManaged(&outlier, unified_size * sizeof(Data));
    hipMallocManaged(&quant, unified_size * sizeof(Quant));

    Data* outlier = data;

    dummy<<<512, 512>>>();
    HANDLE_ERROR(hipDeviceSynchronize());

    auto n = 1;
    if (argc > 0) n = atoi(argv[1]);

    Test1D(n);
    Test2D(n);
    Test3D(n);

    hipFree(data);
    hipFree(quant);
    hipFree(outlier);

    hipDeviceReset();

    return 0;
}
