/**
 * @file test_l2_histsp.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.4
 * @date 2023-05-20
 *
 * (C) 2023 by Indiana University, Argonne National Laboratory
 *
 */

#include <cstdio>
#include <iostream>
#include <random>

#include "kernel2/detail2/hist_sp.inl"

using std::cout;
using std::endl;

using T = uint32_t;
using FQ = uint32_t;

constexpr auto R = 2;
constexpr auto K = 2 * R + 1;

float dist1[] = {0.01, 0.09, 0.8, 0.09, 0.01};
float dist2[] = {0.01, 0.04, 0.9, 0.04, 0.01};
float dist3[] = {0.005, 0.015, 0.96, 0.015, 0.005};

void gen_symetric_dist(
    T *in, size_t inlen, float dist[], int distlen = 5, int offset = 512)
{
  cout << "offset: " << offset << endl;

  auto R = (distlen - 1) / 2;

  std::random_device rd;   // a seed source for the random number engine
  std::mt19937 gen(rd());  // mersenne_twister_engine seeded with rd()
  std::uniform_int_distribution<> distrib(0, inlen);

  for (auto _ = 0; _ < inlen; _++) { in[_] = offset; }
  for (auto i = 0; i < distlen; i++) {
    if (i - R == 0)
      continue;
    else {
      auto N = (int)(inlen * dist[i]);
      auto sym = (i - R) + offset;
      printf("sym: %d, num: %d\n", sym, N);
      for (auto _ = 0; _ < N; _++) {
        auto loc = distrib(gen);
        in[loc] = sym;
      }
    }
  }
}

template <int OUTLEN = 1024, int CHUNK = 32768, int NWARP = 8>
int f_histsp_kernel(size_t inlen, float gen_dist[], int distlen = K)
{
  T *in, *hin;
  FQ *out, *hout;

  hipHostMalloc(&hin, sizeof(T) * inlen);
  memset(hin, 0, sizeof(T) * inlen);
  hipMalloc(&in, sizeof(T) * inlen);
  hipMemset(in, 0, sizeof(T) * inlen);

  hipHostMalloc(&hout, sizeof(FQ) * OUTLEN);
  memset(hout, 0, sizeof(T) * OUTLEN);
  hipMalloc(&out, sizeof(T) * OUTLEN);
  hipMemset(out, 0, sizeof(T) * OUTLEN);

  // setup using randgen
  gen_symetric_dist(hin, inlen, gen_dist, distlen, OUTLEN / 2);

  // for (auto i = 0; i < inlen; i++) cout << hin[i] << "\t";
  // cout << endl;

  hipMemcpy(in, hin, sizeof(T) * inlen, hipMemcpyHostToDevice);

  constexpr auto NTREAD = 32 * NWARP;

  histsp_multiwarp<T, NWARP, CHUNK, FQ>
      <<<(inlen - 1) / CHUNK + 1, NTREAD, sizeof(FQ) * OUTLEN>>>(
          in, inlen, out, OUTLEN, OUTLEN / 2);

  hipDeviceSynchronize();

  // check for error
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  return 0;
}

int main()
{
  constexpr auto r = 64;
  constexpr auto NSYM = r * 2;

  auto inlen = 500 * 500 * 100;
  f_histsp_kernel<NSYM, 16384, 1>(inlen, dist3);
  f_histsp_kernel<NSYM, 16384, 2>(inlen, dist3);
  f_histsp_kernel<NSYM, 16384, 4>(inlen, dist3);
  f_histsp_kernel<NSYM, 16384, 8>(inlen, dist3);
  f_histsp_kernel<NSYM, 16384, 16>(inlen, dist3);
  f_histsp_kernel<NSYM, 16384, 32>(inlen, dist3);

  f_histsp_kernel<NSYM, 32768, 1>(inlen, dist3);
  f_histsp_kernel<NSYM, 32768, 2>(inlen, dist3);
  f_histsp_kernel<NSYM, 32768, 4>(inlen, dist3);
  f_histsp_kernel<NSYM, 32768, 8>(inlen, dist3);
  f_histsp_kernel<NSYM, 32768, 16>(inlen, dist3);
  f_histsp_kernel<NSYM, 32768, 32>(inlen, dist3);

  f_histsp_kernel<NSYM, 65536, 1>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536, 2>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536, 4>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536, 8>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536, 16>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536, 32>(inlen, dist3);

  f_histsp_kernel<NSYM, 65536 * 2, 1>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536 * 2, 2>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536 * 2, 4>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536 * 2, 8>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536 * 2, 16>(inlen, dist3);
  f_histsp_kernel<NSYM, 65536 * 2, 32>(inlen, dist3);

  return 0;
}