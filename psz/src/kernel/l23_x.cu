/**
 * @file l23.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-11-01
 *
 * (C) 2022 by Indiana University, Argonne National Laboratory
 *
 */

#include <hip/hip_runtime.h>

#include "cusz/type.h"
#include "detail/l23_x.cuhip.inl"
#include "kernel/lrz.hh"
#include "utils/err.hh"
#include "utils/timer.hh"

#define L23X_LAUNCH_KERNEL                                                 \
  if (d == 1) {                                                            \
    psz::KERNEL_CUHIP_x_lorenzo_1d1l<                               \
        T, E, T, x_lorenzo<1>::tile.x, x_lorenzo<1>::sequentiality.x>      \
        <<<x_lorenzo<1>::thread_grid(len3), x_lorenzo<1>::thread_block, 0, \
           (hipStream_t)stream>>>(                                        \
            eq, outlier, len3, leap3, radius, ebx2, xdata);                \
  }                                                                        \
  else if (d == 2) {                                                       \
    psz::KERNEL_CUHIP_x_lorenzo_2d1l<T, E, T>                       \
        <<<x_lorenzo<2>::thread_grid(len3), x_lorenzo<2>::thread_block, 0, \
           (hipStream_t)stream>>>(                                        \
            eq, outlier, len3, leap3, radius, ebx2, xdata);                \
  }                                                                        \
  else if (d == 3) {                                                       \
    psz::KERNEL_CUHIP_x_lorenzo_3d1l<T, E, T>                       \
        <<<x_lorenzo<3>::thread_grid(len3), x_lorenzo<3>::thread_block, 0, \
           (hipStream_t)stream>>>(                                        \
            eq, outlier, len3, leap3, radius, ebx2, xdata);                \
  }

template <typename T, typename E, psz_timing_mode TIMING>
pszerror pszcxx_reverse_predict_lorenzo__internal(
    E* eq, dim3 const len3, T* outlier, f8 const eb, int const radius,
    T* xdata, f4* time_elapsed, void* stream)
{
  using namespace psz::kernelconfig;

  // error bound
  auto ebx2 = eb * 2, ebx2_r = 1 / ebx2;
  auto leap3 = dim3(1, len3.x, len3.x * len3.y);
  auto d = lorenzo_utils::ndim(len3);

  if constexpr (TIMING == CPU_BARRIER_AND_TIMING) {
    CREATE_GPUEVENT_PAIR;
    START_GPUEVENT_RECORDING((hipStream_t)stream);

    L23X_LAUNCH_KERNEL;

    STOP_GPUEVENT_RECORDING((hipStream_t)stream);
    CHECK_GPU(hipStreamSynchronize((hipStream_t)stream));
    TIME_ELAPSED_GPUEVENT(time_elapsed);
    DESTROY_GPUEVENT_PAIR;
  }
  else if constexpr (TIMING == CPU_BARRIER) {
    L23X_LAUNCH_KERNEL;
    CHECK_GPU(hipStreamSynchronize((hipStream_t)stream));
  }
  else if constexpr (TIMING == GPU_AUTOMONY) {
    L23X_LAUNCH_KERNEL;
  }
  else {
    throw std::runtime_error(
        "[2403] fail on purpose; show now run into this branch.");
  }

  return CUSZ_SUCCESS;
}

#define L23X_INIT(T, E, TIMING)                                             \
  template pszerror pszcxx_reverse_predict_lorenzo__internal<T, E, TIMING>( \
      E * eq, dim3 const len3, T* outlier, f8 const eb, int const radius,  \
      T* xdata, f4* time_elapsed, void* stream);

L23X_INIT(f4, u4, CPU_BARRIER_AND_TIMING);
L23X_INIT(f8, u4, CPU_BARRIER_AND_TIMING);
L23X_INIT(f4, u4, CPU_BARRIER);
L23X_INIT(f8, u4, CPU_BARRIER);
L23X_INIT(f4, u4, GPU_AUTOMONY);
L23X_INIT(f8, u4, GPU_AUTOMONY);

L23X_INIT(f4, u2, CPU_BARRIER_AND_TIMING);
L23X_INIT(f8, u2, CPU_BARRIER_AND_TIMING);
L23X_INIT(f4, u2, CPU_BARRIER);
L23X_INIT(f8, u2, CPU_BARRIER);
L23X_INIT(f4, u2, GPU_AUTOMONY);
L23X_INIT(f8, u2, GPU_AUTOMONY);

#undef L23X_INIT
#undef L23X_LAUNCH_KERNEL
