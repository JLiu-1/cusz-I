#include "hip/hip_runtime.h"
#include <cstdint>
#include <string>
#include <unordered_map>

#include "detail/fzg_c.cuhip.inl"
#include "detail/fzg_x.cuhip.inl"
#include "kernel/fzg_cx.hh"
#include "utils/err/err.cuhip.hh"

fzgpu::config_map const fzgpu::configure_fzgpu(size_t const data_len)
{
  constexpr auto UINT32_BIT_LEN = 32;
  constexpr auto block_size = 16;
  /* align */ auto data_bytes = data_len * 2;  // how many bytes of data
  /* align */ data_bytes = (data_bytes - 1) / 4096 + 1;
  /* align */ data_bytes *= 4096;
  auto pad_len = data_bytes / 2;
  int data_chunk_size =
      data_bytes % (block_size * UINT32_BIT_LEN) == 0
          ? data_bytes / (block_size * UINT32_BIT_LEN)
          : int(data_bytes / (block_size * UINT32_BIT_LEN)) + 1;

  return config_map{
      {"len", data_len},          {"bytes", data_len * sizeof(float)},
      {"pad_len", pad_len},       {"chunk_size", data_chunk_size},
      {"data_bytes", data_bytes}, {"grid_x", floor(data_bytes / 4096)}};
}

fzgpu::Buf::Buf(config_map const* _config, bool _verifiy_on) :
    verify_on(_verifiy_on), config(_config)
{
  auto grid_x = config->at("grid_x");
  auto pad_len = config->at("pad_len");
  auto chunk_size = config->at("chunk_size");
  auto len = config->at("len");

  CHECK_GPU(hipHostMalloc(&h_in_data, sizeof(uint16_t) * len));
  CHECK_GPU(hipMalloc(&d_in_data, sizeof(uint16_t) * pad_len));

  CHECK_GPU(hipMalloc(&d_comp_out, sizeof(uint16_t) * len));
  CHECK_GPU(hipMalloc(&d_bitflag_array, sizeof(uint32_t) * chunk_size));

  CHECK_GPU(hipHostMalloc(&h_out_data, sizeof(uint16_t) * len));
  CHECK_GPU(hipMalloc(&d_out_data, sizeof(uint16_t) * pad_len));

  CHECK_GPU(hipMalloc(&d_offset_counter, sizeof(uint32_t)));
  CHECK_GPU(hipMalloc(&d_start_pos, sizeof(uint32_t) * grid_x));
  CHECK_GPU(hipMalloc(&d_comp_len, sizeof(uint32_t) * grid_x));

  CHECK_GPU(hipMemset(d_in_data, 0, sizeof(uint16_t) * len));
  CHECK_GPU(hipMemset(d_out_data, 0, sizeof(uint16_t) * len));
  CHECK_GPU(hipMemset(d_bitflag_array, 0, sizeof(uint32_t) * chunk_size));
  CHECK_GPU(hipMemset(d_offset_counter, 0, sizeof(uint32_t)));
  CHECK_GPU(hipMemset(d_start_pos, 0, sizeof(uint32_t) * grid_x));
  CHECK_GPU(hipMemset(d_comp_len, 0, sizeof(uint32_t) * grid_x));
}

fzgpu::Buf::~Buf()
{
  CHECK_GPU(hipHostFree(h_in_data));
  CHECK_GPU(hipFree(d_in_data));

  CHECK_GPU(hipFree(d_out_data));
  CHECK_GPU(hipHostFree(h_out_data));

  CHECK_GPU(hipFree(d_signum));

  CHECK_GPU(hipFree(d_comp_out));
  CHECK_GPU(hipFree(d_bitflag_array));

  CHECK_GPU(hipFree(d_offset_counter));
  CHECK_GPU(hipFree(d_start_pos));
  CHECK_GPU(hipFree(d_comp_len));
}

int fzgpu::cuhip::GPU_FZ_encode(
    uint16_t const* in_data, size_t const data_len,
    uint32_t* space_offset_counter, uint32_t* out_bitflag_array,
    uint32_t* out_start_pos, uint16_t* out_comp, uint32_t* comp_len,
    hipStream_t stream)
{
  auto config = fzgpu::configure_fzgpu(data_len);
  dim3 grid = dim3(config["grid_x"]);
  dim3 block(32, 32);

  fzgpu::KERNEL_CUHIP_fz_fused_encode<<<grid, block, 0, stream>>>(
      (uint32_t*)in_data, config["pad_len"] / 2, space_offset_counter,
      out_bitflag_array, out_start_pos, (uint32_t*)out_comp, comp_len);

  hipStreamSynchronize(stream);

  return 0;
}

int fzgpu::cuhip::GPU_FZ_decode(
    uint16_t const* in_archive, uint32_t* in_bitflag_array,
    uint32_t* in_start_pos, uint16_t* out_decoded, size_t const decoded_len,
    hipStream_t stream)
{
  auto config = fzgpu::configure_fzgpu(decoded_len);
  dim3 grid = dim3(config["grid_x"]);
  dim3 block(32, 32);

  fzgpu::KERNEL_CUHIP_fz_fused_decode<<<grid, block, 0, stream>>>(
      (uint32_t*)in_archive, in_bitflag_array, in_start_pos,
      (uint32_t*)out_decoded, config["pad_len"] / 2);

  hipStreamSynchronize(stream);

  return 0;
}
